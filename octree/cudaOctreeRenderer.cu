#include "hip/hip_runtime.h"
#include "cudaOctreeRenderer.h"

// TODO: How do I separate .h and .cu? Including header files here does not work.
#include "octreeScan.cu"
#include "octreeReduce.cu"

#include <nppdefs.h>
#include <optixu_aabb.h>
#include <optix_math.h>

// TODO: best values?
#define KT 0.1
#define KI 0.9

// !! Assume BIN_COUNT_1D < CUDA_BLOCK_SIZE !!
// !! Assume NUM_WARPS_PER_ROW <= WARP_SIZE !!
// !! Assume WARP_SIZE is power of two !!

// warp size (assume that WARP_SIZE is power of two)
#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

// TODO: set these numbers to right values
#define CUDA_GRID_SIZE_X 15
#define CUDA_GRID_SIZE_Y 12
#define CUDA_GRID_SIZE_Z 1
#define CUDA_GRID_SIZE (CUDA_GRID_SIZE_X * CUDA_GRID_SIZE_Y * CUDA_GRID_SIZE_Z)
#define CUDA_BLOCK_SIZE_X 8
#define CUDA_BLOCK_SIZE_Y 8
#define CUDA_BLOCK_SIZE_Z 4
#define CUDA_BLOCK_SIZE (CUDA_BLOCK_SIZE_X * CUDA_BLOCK_SIZE_Y * CUDA_BLOCK_SIZE_Y)

// #define PARTIAL_SUM_SIZE (CUDA_BLOCK_SIZE_X * CUDA_BLOCK_SIZEY)

// sample
#define SAMPLE_COUNT_X 17
#define SAMPLE_COUNT_Y 17
#define SAMPLE_COUNT_Z 17
#define SAMPLE_COUNT_2D (SAMPLE_COUNT_X * SAMPLE_COUNT_Y)
#define SAMPLE_COUNT_3D (SAMPLE_COUNT_X * SAMPLE_COUNT_Y * SAMPLE_COUNT_Z)
#define BIN_COUNT_X (SAMPLE_COUNT_X - 1)
#define BIN_COUNT_Y (SAMPLE_COUNT_Y - 1)
#define BIN_COUNT_Z (SAMPLE_COUNT_Z - 1)
#define BIN_COUNT (BIN_COUNT_X * BIN_COUNT_Y * BIN_COUNT_Z)

// tree configuration
#define MAX_NUM_NODES (1<<24) // TODO
#define MAX_TREE_DEPTH 10

// global memory allocation
#define HALF_TRI_LIST_SIZE (1<<24) // TODO
#define TRI_LIST_SIZE (HALF_TRI_LIST_SIZE << 1)

#define GLOBAL_BIN_SIZE_PER_BLOCK (BIN_COUNT << 3)
#define GLOBAL_BIN_SIZE (CUDA_GRID_SIZE * GLOBAL_BIN_SIZE_PER_BLOCK)

// local work pool size
#define BATCH_SIZE 32

// local bin size
#define BIN_SIZE (CUDA_BLOCK_SIZE << 3)
#define REORDER_BIN_SIZE (CUDA_BLOCK_SIZE)

// scan
#define SCAN_BUFFER_SIZE (CUDA_BLOCK_SIZE << 1)
#define SCAN_HEADER_SIZE (CUDA_BLOCK_SIZE)

// cost values
#define SAH_COST_SIZE (CUDA_BLOCK_SIZE)

using namespace optix;

namespace oct
{

// TODO: use SOA instead?
struct Node
{
  __device__ __host__
  Node() : id(-1) {
    for (int i=0; i<8; ++i)
      child[i] = -1;
  }
  inline __device__ __host__ 
  void updateAllButChild(int id, int level, int octant, const Aabb& bounds,
                         bool isLeaf, int numTriangles, bool firstHalf)
  {
    this->id = id;
    this->level = level;
    this->octant = octant;
    this->bounds = bounds;
    this->isLeaf = isLeaf;
    this->numTriangles = numTriangles;
    this->firstHalf = firstHalf;
  }
  int id;           // index into the tree
  int level;        // tree level (root: level0)
  int octant;       // octant id (0 to 7)
  Aabb bounds;      // node bounds 
  bool isLeaf;      // indicates this is a leaf node
  int numTriangles; // # triangles bounded by this node
  int child[8];     // index into the tree
  bool firstHalf;   // indicates first half of allocated triangleList space
};

inline __device__
int getLinearThreadId()
{
  return ((blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x);
}

inline __device__
int getNumThreads()
{
  return (blockDim.x * blockDim.y * blockDim.z);
}

inline __device__
int getLocalBinId(int octant, int x, int y, int z)
{
  return (getNumThreads() * octant) + (blockDim.x * blockDim.y * z) + (blockDim.x * y) + x;
}

inline __device__
int getLocalBinId(int x, int y, int z)
{
  return (blockDim.x * blockDim.y * z) + (blockDim.x * y) + x;
}

inline __device__
int getLinearBinId(int x, int y, int z)
{
  return (BIN_COUNT_X * BIN_COUNT_Y * z) + (BIN_COUNT_X * y) + x;
}

inline __device__
int getGlobalBinId(int octant, int x, int y, int z, int blockId=blockIdx.x)
{
  return (blockId * GLOBAL_BIN_SIZE_PER_BLOCK) + (octant * BIN_COUNT) + getLinearBinId(x, y, z);
}

inline __device__
Aabb getBinBounds(const Node& node, int i, int j, int k)
{
  const Aabb& bounds = node.bounds;
  Aabb binBounds;
  float3 diag = bounds[1] - bounds[0];
  float3 step = make_float3(diag.x/BIN_COUNT_X, diag.y/BIN_COUNT_Y, diag.z/BIN_COUNT_Z);
  float3 min = bounds[0] + make_float3(i*step.x, j*step.y, k*step.z);
  binBounds.set(min, min+step);
  return binBounds;
}

inline __device__
float3 getSampleFromBounds(const Node& node, int i, int j, int k)
{
  float3 diag = node.bounds[1] - node.bounds[0];
  float3 step = make_float3(diag.x/(SAMPLE_COUNT_X-1), diag.y/(SAMPLE_COUNT_Y-1), diag.z/(SAMPLE_COUNT_Z-1));
  float3 point = node.bounds[0] + make_float3(i*step.x, j*step.y, k*step.z);
  return point;
}

__device__
int getGlobalTriListBase(int treeLevel, int octant, bool firstHalf)
{
  int partitionSize = HALF_TRI_LIST_SIZE >> (3 * treeLevel);
  int base = (firstHalf==false) * HALF_TRI_LIST_SIZE + partitionSize * octant;
  return base;
}

inline __device__
bool isCornerSample(int x, int y, int z)
{
   bool corner = ((x==0 && y==0) ||
                  (x==SAMPLE_COUNT_X-1 && y==0) ||
                  (x==0 && y==SAMPLE_COUNT_Y-1) ||
                  (x==SAMPLE_COUNT_X-1 && y==SAMPLE_COUNT_Y-1));
   bool bottomCorner = ((z==0) && corner);
   bool topCorner = ((z==SAMPLE_COUNT_Z-1) && corner);
   return (bottomCorner || topCorner);
}

inline __device__
void populateBins(const int3* indices, const float3* vertices,
                  const Node& node, const int* globalTriList,
                  Aabb* triBox, int* bin, int* globalBin)
{
  int numThreads = blockDim.x * blockDim.y * blockDim.z; 

  int binsPerThreadX = (BIN_COUNT_X + blockDim.x - 1) / blockDim.x;
  int binsPerThreadY = (BIN_COUNT_Y + blockDim.y - 1) / blockDim.y;
  int binsPerThreadZ = (BIN_COUNT_Z + blockDim.z - 1) / blockDim.z;

  int trianglesPerThread = (node.numTriangles + numThreads - 1) / numThreads;
  int tid = getLinearThreadId();

  for (int bchunkZ=0; bchunkZ<binsPerThreadZ; ++bchunkZ) {
    for (int bchunkY=0; bchunkY<binsPerThreadY; ++bchunkY) {
      for (int bchunkX=0; bchunkX<binsPerThreadX; ++bchunkX) {

        // warning! # bins > # threads (because of the edges)
        // select the bin
        int binX = blockDim.x * bchunkX + threadIdx.x; 
        int binY = blockDim.y * bchunkY + threadIdx.y;
        int binZ = blockDim.z * bchunkZ + threadIdx.z;
        
        if ((binX<BIN_COUNT_X) && (binY<BIN_COUNT_Y) && (binZ<BIN_COUNT_Z))
        {
          // initialize all bin values to 0 (shared mem)
          for (int octant=0; octant<8; ++octant)
          {
            bin[numThreads * octant + tid] = 0;
          }

          // bin bounds
          Aabb binBox = getBinBounds(node, binX, binY, binZ);
          
          for (int tchunk=0; tchunk<trianglesPerThread; ++tchunk)
          {
            // fetch a triangle and compute its bounding box
            triBox[tid].invalidate();
            int toffset = numThreads * tchunk + tid;
          
            if (toffset < node.numTriangles)
            {
              int base = getGlobalTriListBase(node.level, node.octant, node.firstHalf);
              int triId = globalTriList[base + toffset];

              const int3 vindex = indices[triId];
              triBox[tid].set(vertices[vindex.x], vertices[vindex.y], vertices[vindex.z]);
            }
            __syncthreads();

            // will it be faster to parallelize over traingles?
            // i.e. each thread processes a triangle and iterate over multiple bins.
            // Answer: No! many threads can map to the same bin (i.e. a conflicts).
            //
            // for all triangles in shared mem
            for (int t=0; t<numThreads; ++t)
            {
              Aabb& tbox = triBox[t];
          
              // evaluate the triangle count only if the triangle box falls within the node bounds
              if (tbox.valid() && node.bounds.intersects(tbox))
              {
                // clip the triangle box (i.e. discard the portion outside the node)
                Aabb clippedTbox(fmaxf(node.bounds[0], tbox[0]), fminf(node.bounds[1], tbox[1]));
          
                // finally populate triangle counts for all octants
                for (int octant=0; octant<8; ++octant)
                {
                  // sample one of the triBox points
                  // bottom: sw(0), se(1), nw(2), ne(3)
                  // top   : sw(4), se(5), nw(6), ne(7)
                  int xbit = octant & 0x1;
                  int ybit = (octant >> 1) & 0x1;
                  int zbit = (octant >> 2) & 0x1;
          
                  float3 point = make_float3(clippedTbox[xbit].x, clippedTbox[ybit].y, clippedTbox[zbit].z);
          
                  // do point-binBox intersection test and update the triangle count
                  bin[numThreads * octant + tid] += binBox.contains(point);
                }
              }
            }
          }
        
          // at this point all triangles have been aggregated for the current chunk of bins
          // write the populated triangle counts to global memory
          for (int octant=0; octant<8; ++octant) {
            globalBin[getGlobalBinId(octant, binX, binY, binZ)] = bin[numThreads * octant + tid];
          }
        }
        __syncthreads();
        
        // done counting all triangles for the current chunk of local bins
        // work on the next chunk of global bins
      }
    }
  }
  // done populating all global bins
}

inline __device__
void fetchBins(int octant,
               int xbit,
               int ybit,
               int zbit,
               int bchunkX,
               int bchunkY,
               int bchunkZ,
               int binsPerThreadX,
               int binsPerThreadY,
               int binsPerThreadZ,
               const int* globalBin,
               int* bin)
{
  // source
  // -1 to account for ghost cells
  int sx = bchunkX * (blockDim.x-1) + threadIdx.x + xbit - 1;
  int sy = bchunkY * (blockDim.y-1) + threadIdx.y + ybit - 1;
  int sz = bchunkZ * (blockDim.z-1) + threadIdx.z + zbit - 1;

  // destination
  int dx = threadIdx.x;
  int dy = threadIdx.y;
  int dz = threadIdx.z;

  // populate main and ghost cells 
  bin[getLocalBinId(octant, dx, dy, dz)] = 0;

  if (sx>=0 && sy>=0 && sz>=0 && (sx<BIN_COUNT_X) && (sy<BIN_COUNT_Y) && (sz<BIN_COUNT_Z))
  {
    bin[getLocalBinId(octant, dx, dy, dz)] = globalBin[getGlobalBinId(octant, sx, sy, sz)];
  }
}

inline __device__
void fetchBinsForSampling(const int3& sampleId, const int* globalBin, int* bin)
{
  for (int octant=0; octant<8; ++octant)
  {
    // bottom: sw(0), se(1), nw(2), ne(3)
    // top   : sw(4), se(5), nw(6), ne(7)
    int xbit = octant & 0x1;
    int ybit = (octant >> 1) & 0x1;
    int zbit = (octant >> 2) & 0x1;

    // source
    int sx = sampleId.x + xbit - 1;
    int sy = sampleId.y + ybit - 1;
    int sz = sampleId.z + zbit - 1;
    
    // destination
    int dx = threadIdx.x;
    int dy = threadIdx.y;
    int dz = threadIdx.z;
    
    // populate main and ghost cells 
    bin[getLocalBinId(octant, dx, dy, dz)] = 0;
    
    if (sx<0 || sy<0 || sz<0 || (sx>=SAMPLE_COUNT_X) || (sy>=SAMPLE_COUNT_Y) || (sz>=SAMPLE_COUNT_Z))
      return; 
    
    bin[getLocalBinId(octant, dx, dy, dz)] = globalBin[getGlobalBinId(octant, sx, sy, sz)];
  }
}

#if 0 // assumes N threads, not N-1 threads
inline __device__
void addPartialSums(int dim, int octant,
                    int xbit, int ybit, int zbit,
                    int* bin)
{
  int threadIndex = (dim==0)*threadIdx.x + (dim==1)*threadIdx.y + (dim==2)*threadIdx.z;

  if (threadIndex == 0)
  {
    // source
    int x0 = threadIdx.x + ((xbit==1 && dim==0) * blockDim.x) + ((dim!=0) * (1-xbit));
    int y0 = threadIdx.y + ((ybit==1 && dim==1) * blockDim.y) + ((dim!=1) * (1-ybit));
    int z0 = threadIdx.z + ((zbit==1 && dim==2) * blockDim.z) + ((dim!=2) * (1-zbit));

    // destination
    int x1 = x0 + ((dim==0) * ((xbit==0) - (xbit==1)));
    int y1 = y0 + ((dim==1) * ((ybit==0) - (ybit==1)));
    int z1 = z0 + ((dim==2) * ((zbit==0) - (zbit==1)));

    bin[getLargeBinId(octant, x1, y1, z1)] += bin[getLargeBinId(octant, x0, y0, z0)]
  }
}
#endif

inline __device__
void addPartialSums(int dim, int octant,
                    int xbit, int ybit, int zbit,
                    int* bin)
{
  int threadId = (dim==0)*threadIdx.x + (dim==1)*threadIdx.y + (dim==2)*threadIdx.z;

  if (threadId == 0)
  {
    // source
    int sx = threadIdx.x + (dim==0)*xbit*(blockDim.x-1);
    int sy = threadIdx.y + (dim==1)*ybit*(blockDim.y-1);
    int sz = threadIdx.z + (dim==2)*zbit*(blockDim.z-1);

    // destination
    int dx = threadIdx.x + (dim==0)*(xbit*(blockDim.x-3)+1);
    int dy = threadIdx.y + (dim==1)*(ybit*(blockDim.y-3)+1);
    int dz = threadIdx.z + (dim==2)*(zbit*(blockDim.z-3)+1);

    bin[getLocalBinId(octant, dx, dy, dz)] += bin[getLocalBinId(octant, sx, sy, sz)];
  }
}

inline __device__
void orientBins(int dim, int octant,
                int xbit, int ybit, int zbit,
                const int* bin, int* reorderBin)
{
  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;
  int sizeX = blockDim.x-1;
  int sizeY = blockDim.y-1;
  int sizeZ = blockDim.z-1;
 
  // source
  // int sx = x + (1-xbit);
  // int sy = y + (1-ybit);
  // int sz = z + (1-zbit);

  // destination
  int dx = (dim==0)*(xbit*(sizeX-(x<<1))+x) + (dim==1)*(ybit*(sizeY-(y<<1))+y) + (dim==2)*(zbit*(sizeZ-(z<<1))+z);
  int dy = (dim==0||dim==2)*y + (dim==1)*x;
  int dz = (dim==0||dim==1)*z + (dim==2)*x;

  reorderBin[getLocalBinId(octant, dx, dy, dz)] = bin[getLocalBinId(octant, x, y, z)];
}

inline __device__
void populateScanHeaders(int* scanHeader)
{
  scanHeader[getLocalBinId(threadIdx.x, threadIdx.y, threadIdx.z)] = (threadIdx.x==0);
}

inline __device__
void evaluatePrefixSums(int dim, int octant, int xbit, int ybit, int zbit,
                        const int* din, int* headFlag, int* scratch, int* dout)
{
  int tid = getLinearThreadId();
  int size = blockDim.x * blockDim.y * blockDim.z;

  // source
  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;

  int sizeX = blockDim.x-1;
  // int sizeY = blockDim.y-1; // TODO: this is never used.
  // int sizeZ = blockDim.z-1; // TODO: this is never used.

  int dataIn = din[getLocalBinId(x, y, z)];

  int dataOut = segIncScanBlock(tid, dataIn, scratch, headFlag, size);

  // TODO: check if dy and dz are correct (especially sizeX)
  // destination (inverse the orientations)
  int dx = (dim==0)*(xbit*(sizeX-(x<<1))+x) + (dim==1)*y + (dim==2)*z; 
  int dy = (dim==0||dim==2)*y + (dim==1)*(ybit*(sizeX-(x<<1))+x);
  int dz = (dim==0||dim==1)*z + (dim==2)*(zbit*(sizeX-(x<<1))+x);

  // This is no longer the case.
  // // note: dout is larger than din (due to ghost cells)
  // dx += (1-xbit);
  // dy += (1-ybit);
  // dz += (1-zbit);

  dout[getLocalBinId(octant, dx, dy, dz)] = dataOut;
}

inline __device__
void writeBackTriCounts(int octant, int xbit, int ybit, int zbit,
                        int bchunkX, int bchunkY, int bchunkZ,
                        const int* bin, int* globalBin)
{
  // This is no longer the case.
  // // note: dout is larger than din (due to ghost cells)
  // int sx = threadIdx.x + (1-xbit);
  // int sy = threadIdx.y + (1-ybit);
  // int sz = threadIdx.z + (1-zbit);

  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;

  if (x==0 || y==0 || z==0)
    return;

  int sx = x + (1-(xbit<<1));
  int sy = y + (1-(ybit<<1));
  int sz = z + (1-(zbit<<1));

  int dx = blockDim.x * bchunkX + sx;
  int dy = blockDim.y * bchunkY + sy;
  int dz = blockDim.z * bchunkZ + sz;

  globalBin[getGlobalBinId(octant, dx, dy, dz)] = bin[getLocalBinId(octant, sx, sy, sz)];
}

inline __device__
void accumulateBins(int* bin, int* reorderBin, int* globalBin, int* scanBuffer, int* scanHeader)
{
  int numThreads = getNumThreads();

  // Each denominator subtracts 1 from the block size due to the ghost cell.
  int binsPerThreadX = (BIN_COUNT_X + blockDim.x - 1) / (blockDim.x-1);
  int binsPerThreadY = (BIN_COUNT_Y + blockDim.y - 1) / (blockDim.y-1);
  int binsPerThreadZ = (BIN_COUNT_Z + blockDim.z - 1) / (blockDim.z-1);

  // int tid = getLinearThreadId();

  for (int bchunkZ=0; bchunkZ<binsPerThreadZ; ++bchunkZ) {
    for (int bchunkY=0; bchunkY<binsPerThreadY; ++bchunkY) {
      for (int bchunkX=0; bchunkX<binsPerThreadX; ++bchunkX) {

        for (int octant=0; octant<8; ++octant)
        {

          // bottom: sw(0), se(1), nw(2), ne(3)
          // top   : sw(4), se(5), nw(6), ne(7)
          int xbit = octant & 0x1;
          int ybit = (octant >> 1) & 0x1;
          int zbit = (octant >> 2) & 0x1;

          // 1. fetch counter values from global memory
          fetchBins(octant, xbit, ybit, zbit,
                    bchunkX, bchunkY, bchunkZ,
                    binsPerThreadX, binsPerThreadY, binsPerThreadZ,
                    globalBin, bin);
          __syncthreads();

          for (int dim=0; dim<3; ++dim)
          {
            // We don't need this step because we are using inclusive scan.
            // // 2. add partial sums
            // addPartialSums(dim, octant, xbit, ybit, zbit, bin);
            // __syncthreads();

            // 2. reorder counter values
            orientBins(dim, octant, xbit, ybit, zbit, bin, reorderBin);
            __syncthreads();

            // 3. populate heads
            populateScanHeaders(scanHeader);
            __syncthreads();

            // 4. evaluate prefix sums
            evaluatePrefixSums(dim, octant, xbit, ybit, zbit,
                               reorderBin, scanHeader, scanBuffer, bin);
            __syncthreads();
          }

          // 6. write results back to global memory
          //
          // TODO: better to write back for all octants at once?
          // for now, let's do it on a octant basis.
          //
          writeBackTriCounts(octant, xbit, ybit, zbit,
                             bchunkX, bchunkY, bchunkZ, bin, globalBin);
          __syncthreads();
        }
      }
    }
  }
}

__device__
Aabb getOctantBounds(int octant, const Aabb& node, const float3& point)
{
  // bottom: sw(0), se(1), nw(2), ne(3)
  // top   : sw(4), se(5), nw(6), ne(7)
  int xbit = octant & 0x1;
  int ybit = (octant >> 1) & 0x1;
  int zbit = (octant >> 2) & 0x1;

  float xmin = (xbit==0)*node[0].x + (xbit==1)*point.x;
  float ymin = (ybit==0)*node[0].y + (ybit==1)*point.y;
  float zmin = (zbit==0)*node[0].z + (zbit==1)*point.z;

  float xmax = (xbit==1)*node[1].x + (xbit==0)*point.x;
  float ymax = (ybit==1)*node[1].y + (ybit==0)*point.y;
  float zmax = (zbit==1)*node[1].z + (zbit==0)*point.z;

  float3 pmin = make_float3(xmin, ymin, zmin); 
  float3 pmax = make_float3(xmax, ymax, zmax); 

  Aabb bounds;
  bounds.set(pmin, pmax);

  return bounds;
}

inline __device__
void evaluateSAHCosts(const int3& sampleId, const int* bin, const Aabb& nodeBounds, float* cost)
{
  float sum = 0;
  float3 diag = nodeBounds[1] - nodeBounds[0];
  float3 step = make_float3(diag.x/BIN_COUNT_X, diag.y/BIN_COUNT_Y, diag.z/BIN_COUNT_Z);

  for (int octant=0; octant<8; ++octant)
  {
    // int x = threadIdx.x + xbit;
    // int y = threadIdx.y + ybit;
    // int z = threadIdx.z + zbit;

    float3 sample = nodeBounds[0] + make_float3(sampleId.x * step.x,
                                                sampleId.y * step.y,
                                                sampleId.z * step.z);

    Aabb box = getOctantBounds(octant, nodeBounds, sample);
    float area = box.area();

    int tcount = bin[getLocalBinId(octant, threadIdx.x, threadIdx.y, threadIdx.z)];
    sum += (area * tcount);
  }

  // C = kt + (ki * Sum_i(Ai * Ni)) / A
  cost[getLinearThreadId()] = KT + (KI * sum / nodeBounds.area());;
}

inline __device__
void updateMinCost(int tid, int minTid, float cost, int index,
                   int schunkX, int schunkY, int schunkZ,
                   float* minCost, int* minIndex)
{
  if (tid!=minTid)
    return;

  if (cost<(*minCost))
  {
    *minCost = cost;

    int i = index % blockDim.x;
    int j = (index / blockDim.x) % blockDim.y;
    int k = index / (blockDim.x * blockDim.y);

    minIndex[0] = schunkX * blockDim.x + i;
    minIndex[1] = schunkY * blockDim.y + j;
    minIndex[2] = schunkZ * blockDim.z + k;
  }
}

inline __device__
void sampleSplitPoint(const Node& node, const int* globalBin, int* bin, float* cost, int* index,
                      float* minCost, int* minIndex)
{
  int numThreads = getNumThreads();
  int tid = getLinearThreadId();

  bool minThreadId = (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0);
  if (minThreadId)
  {
    *minCost = NPP_MAXABS_32F; 
    *minIndex = 0;
  }
  __syncthreads();

  int samplesPerThreadX = (SAMPLE_COUNT_X + blockDim.x - 1) / blockDim.x;
  int samplesPerThreadY = (SAMPLE_COUNT_Y + blockDim.y - 1) / blockDim.y;
  int samplesPerThreadZ = (SAMPLE_COUNT_Z + blockDim.z - 1) / blockDim.z;

  for (int schunkZ=0; schunkZ<samplesPerThreadZ; ++schunkZ) {
    for (int schunkY=0; schunkY<samplesPerThreadY; ++schunkY) {
      for (int schunkX=0; schunkX<samplesPerThreadX; ++schunkX) {

        int3 sampleId = make_int3(schunkX * blockDim.x + threadIdx.x,
                                  schunkX * blockDim.y + threadIdx.y,
                                  schunkX * blockDim.z + threadIdx.z);

        // 1. fetch prefix sums from global memory for sampling
        fetchBinsForSampling(sampleId, globalBin, bin);
        __syncthreads();
         
        // 2. evaluate the cost function
        evaluateSAHCosts(sampleId, bin, node.bounds, cost);
        __syncthreads();

        // 3. evaluate the minimum cost value
        minReduceBlock(tid, numThreads, cost, index);
        __syncthreads();

        // 4. save min cost and index
        updateMinCost(tid, minThreadId, cost[0], index[0],
                      schunkX, schunkY, schunkZ, minCost, minIndex);
        __syncthreads();
      }
    }
  }
}

inline __device__
bool determineLeaf(int treeLevel)
{
  return (treeLevel >= MAX_TREE_DEPTH);
}

inline __device__
void createChildNodes(Node& node,
                      const int* minIndex, // smem, size=3
                      int* globalTriList,
                      const int3* indices,
                      const float3* vertices,
                      int* scanBuf, // smem, size=thread block
                      int* partialSum, // smem, size=8, one for each octant
                      Node* globalTree,
                      int* outPoolIndex,
                      int* numOutputNodes)
{
  int tid = getLinearThreadId();
  int numThreads = getNumThreads();

  // set the node to a leaf if the sample is at a corner.
  if (isCornerSample(minIndex[0], minIndex[1], minIndex[2]))
  {
    if (tid==numThreads-1)
    {
      // TODO: we only need to udpate isLeaf, not all member variables.
      node.isLeaf = true;
      globalTree[node.id] = node;
    }
    return;
  }

  // initialize partial sums
  if (tid<8)
  {
    partialSum[tid] = 0;
  }
 
  int trianglesPerThread = (node.numTriangles + numThreads - 1) / numThreads;

  for (int tchunk=0; tchunk<trianglesPerThread; ++tchunk)
  {
    int triId = -1;
    Aabb triBox;
    triBox.invalidate();

    const int toffset = numThreads * tchunk + tid;

    // 1. evaluate my triangle bounds
    if (toffset < node.numTriangles)
    {
      int base = getGlobalTriListBase(node.level, node.octant, node.firstHalf);
      int triId = globalTriList[base + toffset];
      const int3 vindex = indices[triId];
      triBox.set(vertices[vindex.x], vertices[vindex.y], vertices[vindex.z]);
    }
    // __syncthreads();

    // 2. evaluate the point to split
    float3 point = getSampleFromBounds(node, minIndex[0], minIndex[1], minIndex[2]);

    for (int o=0; o<8; ++o) // for each octant o
    {
      // 3. evaluate the octant bound
      Aabb octant = getOctantBounds(o, node.bounds, point);

      if (octant.volume() > 0)
      {
        const Aabb& tbox = triBox;
        
        // TODO: check if box-to-box test is a good solution.
        //       perhaps, box-to-triangle could be a better method?
        // 4. do a trinagle-octant intersection test
        int triHit = (tbox.valid() && octant.intersects(tbox));
        __syncthreads();
        
        // 5. scan over triHit to get offset into globalTriList
        int triListOffset = incScanBlock(tid, triHit, scanBuf, numThreads);
        __syncthreads();
        
        // TODO: for now, let's assume equal distribution of triangles
        // TODO: should it be faster to buffer triangle IDs into shared memeory
        //       and write them back to global memory later at once?
        //
        // 6. update globalTriList with triangle IDs if they belong to the octant
        if (triHit)
        {
          int base = getGlobalTriListBase((node.level+1), o, (!node.firstHalf));
          globalTriList[partialSum[o] + base + triListOffset - 1] = triId;
        }

        if (tid==numThreads-1)
        {
          // 7. update the partial sum
          int base = getGlobalTriListBase((node.level+1), o, (!node.firstHalf));
          partialSum[o] = partialSum[o] + base + triListOffset;

          // TODO: check if outPoolIndex is within the bound.
          if (tchunk==trianglesPerThread-1)
          {
            // 8. update # of output nodes as well as the child pointer
            int outNodeCount = atomicAdd(numOutputNodes, 1);

            int childId = atomicAdd(outPoolIndex, 1);
            node.child[o] = childId;
            
            // 9. create a child node
            Node cn;
            bool isLeaf = determineLeaf(node.level+1); 
            cn.updateAllButChild(childId, (node.level+1), o, octant, isLeaf, triListOffset, (!node.firstHalf));
            globalTree[childId] = cn;
          }
        }
      } // if (octant.volume() > 0)

      // TODO: do we need to update all member variables at once?
      //       i.e. not all of them need to be udpated.
      //       we only need to update child[8], 32B.
      // 10. update the parent node's child pointers
      if (tid==numThreads-1 && tchunk==trianglesPerThread-1 && o==7)
      {
        globalTree[node.id] = node; 
      }
    } // for (int o=0; o<8; ++o) // for each octant o
  } // for (int tchunk=0; tchunk<trianglesPerThread; ++tchunk)
}

inline __device__
void buildOctree(const int3* indices, const float3* vertices,
                 Node& node, Node* globalTree, int* globalTriList,
                 int* outPoolIndex, int* numOutputNodes,
                 Aabb* triBox, int* bin, int* reorderBin, int* globalBin,
                 int* scanBuffer, int* scanHeader, float* sahCost,
                 float* minCost, int* minIndex)
{
  if (node.isLeaf)
    return;

  populateBins(indices, vertices, node, globalTriList, triBox, bin, globalBin);
  __syncthreads();

  accumulateBins(bin, reorderBin, globalBin, scanBuffer, scanHeader);
  __syncthreads();

  // note: reorderBin is also used to maintain thread indices
  sampleSplitPoint(node, globalBin, bin, sahCost, reorderBin, minCost, minIndex);
  __syncthreads();

  createChildNodes(node, minIndex, globalTriList, indices, vertices,
                   scanBuffer, scanHeader, globalTree,
                   outPoolIndex, numOutputNodes); 
  // __syncthreads();
}

__global__
void buildKernel(const int3* indices, const float3* vertices,
                 const int* numInputNodes, int* numOutputNodes,
                 int* inPoolIndex, int* outPoolIndex,
                 Node* globalTree, int* globalTriList, int* globalBin)
{
  // for maintaining work pools
  __shared__ Node localPool[BATCH_SIZE];
  __shared__ int localPoolSize;
  __shared__ int inPoolSize;
  __shared__ int baseIdx;
  __shared__ int localPoolIdx;

  // for counting triangles
  __shared__ Aabb triBox[CUDA_BLOCK_SIZE];
  __shared__ int bin[BIN_SIZE]; // triangle counts
  __shared__ int reorderBin[REORDER_BIN_SIZE];
  __shared__ int scanBuffer[SCAN_BUFFER_SIZE];
  __shared__ int scanHeader[SCAN_HEADER_SIZE]; __shared__ float sahCost[SAH_COST_SIZE];

  // TODO: should we just use registers for minCost and minIndex?
  __shared__ float minCost;
  __shared__ int minIndex[3];

  int tid = getLinearThreadId();
  bool fetcherThread = (tid==0);

  if (fetcherThread)
  {
    localPoolSize = 0;
    inPoolSize = *numInputNodes;
  }
  __syncthreads();

  while(true)
  {
    // fetch work if local pool is empty
    if (localPoolSize == 0)
    {
      if (fetcherThread)
      {
        baseIdx = atomicAdd(inPoolIndex, BATCH_SIZE);
        localPoolIdx = 0;
        localPoolSize = BATCH_SIZE;
      }
      __syncthreads();

      // exit if no more work left
      if (baseIdx >= inPoolSize)
        return;

      // fetch work from the work pool in global memory
      if (tid < BATCH_SIZE)
      {
        localPool[tid] = Node();
        int index = baseIdx + tid;

        // fetch work if within the range
        if (index < inPoolSize)
        {
          localPool[tid] = globalTree[index];
        }
      }
    }
    __syncthreads();

    // work is valid if nodeID is non-negative
    if (localPool[localPoolIdx].id >= 0)
    {
      buildOctree(indices, vertices,
                  localPool[localPoolIdx], globalTree, globalTriList,
                  outPoolIndex, numOutputNodes,
                  triBox, bin, reorderBin, globalBin, scanBuffer, scanHeader, sahCost,
                  &minCost, minIndex);
    }

    // next work to process
    if (fetcherThread)
    {
      ++localPoolIdx;
      --localPoolSize;
    }
    __syncthreads();
  }
}

inline __device__
void updateClosest(const Hit& isect, Hit& closest)
{
  closest.t = isect.t;
  closest.triId = isect.triId; closest.u = isect.u;
  closest.v = isect.v;
}

inline __device__
void updateHitBuffer(const Hit& closest, Hit* hitBuf)
{
  hitBuf->t = closest.t;
  hitBuf->triId = closest.triId;
  hitBuf->u = closest.u;
  hitBuf->v = closest.v;
}

CUDAOctreeRenderer::CUDAOctreeRenderer(const ConfigLoader& config)
: RTPSimpleRenderer(config) {}

void CUDAOctreeRenderer::render()
{
  int3* d_indices;
  float3* d_vertices;
  //int rounded_length = nextPow2(length);

  CHK_CUDA(hipMalloc((void**)&d_indices, scene.numTriangles * sizeof(int3)));
  CHK_CUDA(hipMalloc((void**)&d_vertices, scene.numTriangles * sizeof(float3)));

  CHK_CUDA(hipMemcpy(d_indices, scene.indices,
                      scene.numTriangles * sizeof(int3), hipMemcpyHostToDevice));
  CHK_CUDA(hipMemcpy(d_vertices, scene.vertices,
                      scene.numTriangles * sizeof(float3), hipMemcpyHostToDevice));

  // maintain # nodes to process and # newly created nodes
  int* d_numInputNodes;
  int* d_numOutputNodes;
  CHK_CUDA(hipMalloc((void**)&d_numInputNodes, sizeof(int)));
  CHK_CUDA(hipMalloc((void**)&d_numOutputNodes, sizeof(int)));

  // maintain pool indices
  int h_inPoolIndex = 0;
  int h_outPoolIndex = 1;
  int* d_inPoolIndex;
  int* d_outPoolIndex;
  CHK_CUDA(hipMalloc((void**)&d_inPoolIndex, sizeof(int)));
  CHK_CUDA(hipMalloc((void**)&d_outPoolIndex, sizeof(int)));
  CHK_CUDA(hipMemcpy(d_inPoolIndex, &h_inPoolIndex, sizeof(int), hipMemcpyHostToDevice));
  CHK_CUDA(hipMemcpy(d_outPoolIndex, &h_outPoolIndex, sizeof(int), hipMemcpyHostToDevice));

  // create root node
  Node* h_octree = new Node;
  h_octree->id = 0;
  h_octree->level = 0;
  h_octree->octant = 0;
  h_octree->bounds = Aabb(scene.bbmin, scene.bbmax);
  h_octree->isLeaf = false;
  h_octree->numTriangles = scene.numTriangles;
  for (int o=0; o<8; ++o)
  {
    h_octree->child[o] = -1;
  }
  h_octree->firstHalf = true;
  Node* d_tree;
  CHK_CUDA(hipMalloc((void**)&d_tree, MAX_NUM_NODES * sizeof(Node)));
  CHK_CUDA(hipMemcpy(d_tree, h_octree, sizeof(Node), hipMemcpyHostToDevice));

  // create triangle lists
  int* d_triList;
  CHK_CUDA(hipMalloc((void**)&d_triList, TRI_LIST_SIZE * sizeof(int)));

  // bins to store triangle counts
  int* d_bin;
  CHK_CUDA(hipMalloc((void**)&d_bin, GLOBAL_BIN_SIZE * sizeof(int)));

  // build(d_indices, d_vertices, d_workPoolA, d_workPoolB, d_bin);
  build(d_indices, d_vertices, d_numInputNodes, d_numOutputNodes,
        d_inPoolIndex, d_outPoolIndex, d_tree, d_triList, d_bin);
  // trace(d_indices, d_vertices);

  hipFree(d_indices);
  hipFree(d_vertices);
  hipFree(d_numInputNodes);
  hipFree(d_numOutputNodes);
  hipFree(d_inPoolIndex);
  hipFree(d_outPoolIndex);
  hipFree(d_tree);
  hipFree(d_bin);

  delete [] h_octree;
}

void CUDAOctreeRenderer::build(const int3* indices, const float3* vertices,
                               int* d_numInputNodes, int* d_numOutputNodes,
                               int* d_inPoolIndex, int* d_outPoolIndex,
                               Node* d_tree, int* d_triList, int* d_bin)
{
  // TODO: Use Occupancy APIs to determine grid and block sizes
  // supported for CUDA 6.5 and above
  dim3 gridDim(CUDA_GRID_SIZE);
  dim3 blockDim(CUDA_BLOCK_SIZE_X, CUDA_BLOCK_SIZE_Y, CUDA_BLOCK_SIZE_Z);

  int h_workLeft = 1;
  while(h_workLeft)
  {
    // TODO: is there any better way than this? (i.e. not transferring values between kernel calls?)
    // but this should take a small portion of the whole build time
    // since only (4B * 3 * d) Bytes of data transfer involved for the whole process, where d = tree level.
    int h_numOutputNodes = 0; 
    CHK_CUDA(hipMemcpy(d_numInputNodes, &h_workLeft, sizeof(int), hipMemcpyHostToDevice));
    CHK_CUDA(hipMemcpy(d_numOutputNodes, &h_numOutputNodes, sizeof(int), hipMemcpyHostToDevice));

    buildKernel<<<gridDim, blockDim>>>(indices, vertices,
                                       d_numInputNodes, d_numOutputNodes,
                                       d_inPoolIndex, d_outPoolIndex,
                                       d_tree, d_triList, d_bin);
    hipDeviceSynchronize();
    CHK_CUDA(hipMemcpy(&h_workLeft, d_numOutputNodes, sizeof(int), hipMemcpyDeviceToHost));
  }
}

}
