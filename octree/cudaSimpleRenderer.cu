#include "hip/hip_runtime.h"
#include "cudaSimpleRenderer.h"
#include <nppdefs.h>

#define kEpsilon 1e-18

inline __device__ void updateClosest(const Hit& isect, Hit& closest)
{
  closest.t = isect.t;
  closest.triId = isect.triId; closest.u = isect.u;
  closest.v = isect.v;
}

inline __device__ void updateHitBuffer(const Hit& closest, Hit* hitBuf)
{
  hitBuf->t = closest.t;
  hitBuf->triId = closest.triId;
  hitBuf->u = closest.u;
  hitBuf->v = closest.v;
}

inline __device__ bool intersect(const Ray& ray, const int3* indices, const float3* vertices, const int triId, Hit& isect) {
  const int3 tri = indices[triId];
  const float3 a = vertices[tri.x];
  const float3 b = vertices[tri.y];
  const float3 c = vertices[tri.z];
  const float3 e1 = b - a;
  const float3 e2 = c - a;
  const float3 p_vec = cross(ray.dir, e2);
  float det = dot(e1, p_vec);
  if (det > -kEpsilon && det < kEpsilon)
    return false;
  float inv_det = 1.0f / det;
  float3 t_vec = ray.origin - a;
  float3 q_vec = cross(t_vec, e1);
  float t = dot(e2, q_vec) * inv_det;
  // Do not allow ray origin in front of triangle
  if (t < 0.0f)
    return false;
  float u = dot(t_vec, p_vec) * inv_det;
  if (u < 0.0f || u > 1.0f)
    return false;
  float v = dot(ray.dir, q_vec) * inv_det;
  if (v < 0.0f || u + v > 1.0f)
    return false;

  isect.t = t;
  isect.triId = triId;
  isect.u = u;
  isect.v = v;
  return true;
}

__global__ void simpleTraceKernel(const Ray* rays,
                                  const int3* indices, const float3* vertices,
                                  const int rayCount, const int triCount,
                                  Hit* hits) {
  int rayIdx = threadIdx.x + blockIdx.x*blockDim.x;

  if (rayIdx >= rayCount) {
    return;
  }
  
  Hit closest;
  closest.t = NPP_MAXABS_32F;
  closest.triId = -1;
  const Ray& ray = *(rays + rayIdx);
  for (int t=0; t<triCount; ++t) { // triangles
    Hit isect;
    if (intersect(ray, indices, vertices, t, isect)) {
      //printf("intersect!\n");
      if (isect.t < closest.t) {
        updateClosest(isect, closest);
      }
    }
  }
  updateHitBuffer(closest, (hits+rayIdx));
}

CUDASimpleRenderer::CUDASimpleRenderer(const ConfigLoader& config)
: RTPSimpleRenderer(config) {}

void CUDASimpleRenderer::render() {
  int3 *d_indices;
  float3 *d_vertices;
  //int rounded_length = nextPow2(length);

  CHK_CUDA(hipMalloc((void **)&d_indices, scene.numTriangles * sizeof(int3)));
  CHK_CUDA(hipMalloc((void **)&d_vertices, scene.numTriangles * sizeof(float3)));

  CHK_CUDA(hipMemcpy(d_indices, scene.indices, scene.numTriangles * sizeof(int3), hipMemcpyHostToDevice));
  CHK_CUDA(hipMemcpy(d_vertices, scene.vertices, scene.numTriangles * sizeof(float3), hipMemcpyHostToDevice));

  simpleTraceOnDevice(d_indices, d_vertices);

  hipFree(d_indices);
  hipFree(d_vertices);
}

void CUDASimpleRenderer::simpleTraceOnDevice(const int3* indices, const float3* vertices) {

  const int numThreadsPerBlock = 256;
  const int numBlocks = (rayBuffer.count() + numThreadsPerBlock - 1) / numThreadsPerBlock;
  
  simpleTraceKernel<<<numBlocks, numThreadsPerBlock>>>(rayBuffer.ptr(),
                                                       indices, vertices, rayBuffer.count(),
                                                       scene.numTriangles,
                                                       hitBuffer.ptr());
}
