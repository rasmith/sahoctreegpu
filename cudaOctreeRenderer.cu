#include "hip/hip_runtime.h"
#include "cudaOctreeRenderer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nppdefs.h>
#include <float.h>
#include <sys/time.h>

#include "log.h"
#include "octree.h"
#include "cuda_math.h"

#define kEpsilon 1e-18

#define USE_PERSISTENT

#define WARP_SIZE 32   // Hardware size of a warp, 32 lanes.
#define WARP_FACTOR 4  // How many warps per block do we want.
#define THREADS_PER_BLOCK (WARP_FACTOR * WARP_SIZE)  // Compute # threads.
#define WARPS_PER_BLOCK WARP_FACTOR
#define REGISTERS_PER_SM (1 << 15)
#define SHARED_MEMORY_PER_SM (1 << 15)
#define MAX_REGISTERS_THREAD 63
#define MIN_BLOCKS \
  ((REGISTERS_PER_SM) / (THREADS_PER_BLOCK * MAX_REGISTERS_THREAD))
#define MAX_SHARED_MEMORY_PER_BLOCK SHARED_MEMORY_PER_SM / MIN_BLOCKS
#define MAX_BLOCKS_PER_DIMENSION 65535
//#define UPDATE_HITS_SOA

#define WARP_LOAD_FACTOR 3  // This is effectively #rays / threads
#define WARP_BATCH_SIZE (WARP_LOAD_FACTOR * WARP_SIZE)  // #rays / warp batch
__device__ int nextRayIndex;

//#define USE_TRACE_KERNEL_LAUNCH_BOUNDS
texture<uint4, 1, hipReadModeElementType> texture_nodes;
texture<float4, 1, hipReadModeElementType> texture_vertices;
texture<int4, 1, hipReadModeElementType> texture_indices;
texture<uint32_t, 1, hipReadModeElementType> texture_references;

#define GET_RAY_ORIGIN(rays, width, pitch, i)                               \
  *(reinterpret_cast<const float4*>(reinterpret_cast<const char*>((rays)) + \
                                    ((i) / (width)) * (pitch)) +            \
    2 * ((i) % (width)))

#define GET_RAY_DIRECTION(rays, width, pitch, i)                            \
  *(reinterpret_cast<const float4*>(reinterpret_cast<const char*>((rays)) + \
                                    ((i) / (width)) * (pitch)) +            \
    2 * ((i) % (width)) + 1)

#define SET_HIT(hits, width, pitch, i, x)                    \
  *(reinterpret_cast<Hit*>(reinterpret_cast<char*>((hits)) + \
                           ((i) / (width)) * (pitch)) +      \
    ((i) % (width))) = x

namespace oct {
template <uint32_t N>
__host__ __device__ inline uint32_t lg2() {
  return ((N >> 1) != 0) + lg2<(N >> 1)>();
}

template <>
__host__ __device__ inline uint32_t lg2<0>() {
  return 0;
}

template <>
__host__ __device__ inline uint32_t lg2<1>() {
  return 0;
}

struct Ray4 {
  float4 origin;
  float4 dir;
};

struct Aabb4 {
  float4 min;
  float4 max;
};

std::ostream& operator<<(std::ostream& os, const Ray& r) {
  float3 origin = make_float3(r.ox, r.oy, r.oz);
  float3 dir = make_float3(r.dx, r.dy, r.dz);
  os << "o = " << origin << " tmin = " << r.tmin << " d = " << dir
     << " tmax = " << r.tmax;
  return os;
}

inline __host__ __device__ float4 cross(const float4& a, const float4& b) {
  return make_float4(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
                     a.x * b.y - a.y * b.x, 0.0f);
}

inline __host__ __device__ float dot43(const float4& a, const float4& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

template <typename SourceType, typename DestinationType>
void __host__ __device__ assign(const SourceType& source,
                                DestinationType* dest) {}

template <>
void __host__ __device__ assign<float3, float4>(const float3& source,
                                                float4* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
  dest->w = 0.0f;
}

template <>
void __host__ __device__ assign<int3, int4>(const int3& source, int4* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
  dest->w = 0.0f;
}

template <>
void __host__ __device__ assign<float4, float3>(const float4& source,
                                                float3* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
}

template <>
void __host__ __device__ assign<int4, int3>(const int4& source, int3* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
}

struct timespec getRealTime() {
  struct timespec ts;
#ifdef __FreeBSD__
  clock_gettime(CLOCK_MONOTONIC, &ts);  // Works on FreeBSD
#else
  clock_gettime(CLOCK_REALTIME, &ts);
#endif
  return ts;
}

template <typename T>
inline __device__ __host__ const T* RunTimeSelect(bool condition,
                                                  const T* trueResult,
                                                  const T* falseResult) {
  const uintptr_t c = condition * ~(static_cast<uintptr_t>(0x0));
  return reinterpret_cast<const T*>(
      ((reinterpret_cast<uintptr_t>(trueResult) & c) |
       (reinterpret_cast<uintptr_t>(falseResult) & ~c)));
}

template <typename T>
inline __device__ __host__ void RunTimeAssignIf(bool condition, T* dest,
                                                const T* src) {
  T dummy;
  const uintptr_t c = condition * ~(static_cast<uintptr_t>(0x0));
  *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(dest) & c) |
                         (reinterpret_cast<uintptr_t>(&dummy) & ~c))) = *src;
}

double getTimeDiffMs(const struct timespec& start, const struct timespec& end) {
  // start: X s, A ns
  // end:   Y s, B ns
  // (Y - (X + 1)) * 1000000.0 + B / 1000.0 + 1000000.0 - A / 1000.0
  // = (Y - X) * 1000000.0 - 1000000.0 + B / 1000.0 + 1000000.0 - A / 1000.0
  // = (Y - X) * 1000000.0 + B / 1000.0 - A / 1000.0
  double microsecond_diff = 1000000.0 * (end.tv_sec - start.tv_sec) +
                            end.tv_nsec / 1000.0 - start.tv_nsec / 1000.0;
  return microsecond_diff;
}

#ifdef UPDATE_HITS_SOA
__global__ __launch_bounds__(THREADS_PER_BLOCK,
                             MIN_BLOCKS) void reorderHitsKernel(Hit* hits,
                                                                int numRays) {
  __shared__ Hit localHits[THREADS_PER_BLOCK];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < numRays) {
    float* t_values = reinterpret_cast<float*>(hits + blockIdx.x * blockDim.x);
    int* triIds = reinterpret_cast<int*>(t_values + blockDim.x);
    float* u_values = reinterpret_cast<float*>(triIds + blockDim.x);
    float* v_values = u_values + blockDim.x;
    localHits[threadIdx.x].t = t_values[threadIdx.x];
    localHits[threadIdx.x].triId = triIds[threadIdx.x];
    localHits[threadIdx.x].u = u_values[threadIdx.x];
    localHits[threadIdx.x].v = v_values[threadIdx.x];
  }
  __syncthreads();
  if (tid < numRays) {
    hits[threadIdx.x + blockIdx.x * blockDim.x] = localHits[threadIdx.x];
  }
}
#endif

#define DIVERGENCE_FREE_CHILD_BOUNDS
inline __device__ __host__ Aabb4 getChildBounds(const Aabb4& bounds,
                                                const float4& center,
                                                unsigned char octant) {
  Aabb4 result;
  float4 min = bounds.min;
  float4 max = bounds.max;
#ifdef DIVERGENCE_FREE_CHILD_BOUNDS
  const float4* min_center[2] = {&min, &center};
  const float4* center_max[2] = {&center, &max};
#endif

#ifdef DIVERGENCE_FREE_CHILD_BOUNDS
  unsigned char xBit = (octant >> 0) & 0x1;
  unsigned char yBit = (octant >> 1) & 0x1;
  unsigned char zBit = (octant >> 2) & 0x1;
  min.x = min_center[xBit]->x;
  max.x = center_max[xBit]->x;
  min.y = min_center[yBit]->y;
  max.y = center_max[yBit]->y;
  min.z = min_center[zBit]->z;
  max.z = center_max[zBit]->z;
#else
  min.x = ((octant & (0x1 << 0)) > 0 ? center.x : min.x);
  max.x = ((octant & (0x1 << 0)) > 0 ? max.x : center.x);
  min.y = ((octant & (0x1 << 1)) > 0 ? center.y : min.y);
  max.y = ((octant & (0x1 << 1)) > 0 ? max.y : center.y);
  min.z = ((octant & (0x1 << 2)) > 0 ? center.z : min.z);
  max.z = ((octant & (0x1 << 2)) > 0 ? max.z : center.z);
#endif
  result.min = min;
  result.max = max;
  return result;
}

inline __device__ __host__ bool isValidT(float t, float t_near, float t_far) {
  return !isnan(t) & t < t_far & t >= t_near;
}

template <typename T>
inline __device__ __host__ void exchangeIf(bool condition, T* temp, T* x,
                                           T* y) {
  uintptr_t c = condition;
  c -= 1;
  *temp = *x;
  *x = *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(x) & c) |
                              (reinterpret_cast<uintptr_t>(y) & ~c)));
  *y = *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(temp) & ~c) |
                              (reinterpret_cast<uintptr_t>(y) & c)));
}

template <>
inline __device__ __host__ void exchangeIf<unsigned char>(bool condition,
                                                          unsigned char* temp,
                                                          unsigned char* x,
                                                          unsigned char* y) {
  unsigned char c = condition;
  c -= 1;
  *temp = *x;
  *x = ((((*x) & c) | ((*y) & ~c)));
  *y = (((*temp) & ~c) | ((*y) & c));
}

//#define USE_COALESCED_HIT_UPDATE
inline __device__ __host__ void updateHitBuffer(Hit* closest, Hit* hitBuf) {
#ifdef USE_COALESCED_HIT_UPDATE
  unsigned char* out = reinterpret_cast<unsigned char*>(hitBuf);
  uchar4 c0 = *reinterpret_cast<const uchar4*>(&closest.t);
  uchar4 c4 = *reinterpret_cast<const uchar4*>(&closest.triId);
  uchar4 c8 = *reinterpret_cast<const uchar4*>(&closest.u);
  uchar4 c12 = *reinterpret_cast<const uchar4*>(&closest.v);
  out[0] = c0.x;
  out[1] = c4.y;
  out[2] = c8.z;
  out[3] = c12.w;
#else
  hitBuf->t = closest->t;
  hitBuf->triId = closest->triId;
  hitBuf->u = closest->u;
  hitBuf->v = closest->v;
#endif
}

__device__ __inline__ float min4(float a, float b, float c, float d) {
  return fminf(fminf(fminf(a, b), c), d);
}

__device__ __inline__ float max4(float a, float b, float c, float d) {
  return fmaxf(fmaxf(fmaxf(a, b), c), d);
}

#define DIVERGENCE_FREE_INSTERSECT_AABB
inline __device__ __host__ bool intersectAabb(const float4& origin,
                                              const float4& invDirection,
                                              const Aabb4& bounds, float t0,
                                              float t1, float* tNear,
                                              float* tFar) {
  const float4 localBounds[2] = {bounds.min, bounds.max};
  const unsigned char s[3] = {invDirection.x < 0, invDirection.y < 0,
                              invDirection.z < 0};
#ifdef DIVERGENCE_FREE_INSTERSECT_AABB
  float tN = (localBounds[s[0]].x - origin.x) * invDirection.x;
  float tF = (localBounds[1 - s[0]].x - origin.x) * invDirection.x;
#else
  *tNear = (localBounds[s[0]].x - origin.x) * invDirection.x;
  *tFar = (localBounds[1 - s[0]].x - origin.x) * invDirection.x;
#endif
  float tymin = (localBounds[s[1]].y - origin.y) * invDirection.y;
  float tymax = (localBounds[1 - s[1]].y - origin.y) * invDirection.y;

#ifdef DIVERGENCE_FREE_INSTERSECT_AABB
  tN = max(tN, tymin);
  tF = min(tF, tymax);
#else
  if (*tNear > tymax || tymin > *tFar) return false;
  if (tymin > *tNear) *tNear = tymin;
  if (tymax < *tFar) *tFar = tymax;
#endif

  float tzmin = (localBounds[s[2]].z - origin.z) * invDirection.z;
  float tzmax = (localBounds[1 - s[2]].z - origin.z) * invDirection.z;

#ifdef DIVERGENCE_FREE_INSTERSECT_AABB
  tN = max(tN, tzmin);
  tF = min(tF, tzmax);
#else
  if (*tNear > tzmax || tzmin > *tFar) return false;
  if (tzmin > *tNear) *tNear = tzmin;
  if (tzmax < *tFar) *tFar = tzmax;
#endif

#ifdef DIVERGENCE_FREE_INSTERSECT_AABB
  *tNear = tN;
  *tFar = tF;
  return !(tN > tF);
#else
  return *tNear<t1&& * tFar> t0;
#endif
}

inline __device__ bool intersectAabb2(const float4& origin,
                                      const float4& invDirection,
                                      const Aabb4& bounds, float t0, float t1,
                                      float* tNear, float* tFar) {
  const float4 ood =
      make_float4(origin.x * invDirection.x, origin.y * invDirection.y,
                  origin.z * invDirection.z, 0.0f);
  const float4& min_bounds = bounds.min;
  const float4& max_bounds = bounds.max;
  float4 min_bounds_diff =
      make_float4(min_bounds.x - origin.x, min_bounds.y - origin.y,
                  min_bounds.z - origin.z, 0.0f);
  float4 tmins = make_float4(min_bounds_diff.x * invDirection.x,
                             min_bounds_diff.y * invDirection.y,
                             min_bounds_diff.z * invDirection.z, 0.0f);
  float4 max_bounds_diff =
      make_float4(max_bounds.x - origin.x, max_bounds.y - origin.y,
                  max_bounds.z - origin.z, 0.0f);
  float4 tmaxs = make_float4(max_bounds_diff.x * invDirection.x,
                             max_bounds_diff.y * invDirection.y,
                             max_bounds_diff.z * invDirection.z, 0.0f);
  float tminbox = max4(t0, fminf(tmins.x, tmaxs.x), fminf(tmins.y, tmaxs.y),
                       fminf(tmins.z, tmaxs.z));
  float tmaxbox = min4(t1, fmaxf(tmins.x, tmaxs.x), fmaxf(tmins.y, tmaxs.y),
                       fmaxf(tmins.z, tmaxs.z));
  bool intersect = (tminbox <= tmaxbox);
  *tNear = tminbox;
  *tFar = tmaxbox;
  return intersect;
}

#define DIVERGENCE_FREE_INSTERSECT_TRIANGLE
inline __device__ bool intersectTriangle(const float4& origin,
                                         const float4& dir, const int4* indices,
                                         const float4* vertices, int triId,
                                         Hit& isect, int numTriangles,
                                         int numVertices) {
  const int4 tri = indices[triId];
  /*const int4 tri = tex1Dfetch(texture_indices, triId);*/
  const float4 a = vertices[tri.x];
  const float4 b = vertices[tri.y];
  const float4 c = vertices[tri.z];
  /*const float4 a = tex1Dfetch(texture_vertices, tri.x);*/
  /*const float4 b = tex1Dfetch(texture_vertices, tri.y);*/
  /*const float4 c = tex1Dfetch(texture_vertices, tri.z);*/
  const float4 e1 = b - a;
  const float4 e2 = c - a;
  const float4 pVec =
      make_float4(dir.y * e2.z - dir.z * e2.y, dir.z * e2.x - dir.x * e2.z,
                  dir.x * e2.y - dir.y * e2.x, 0.0f);
  float det = dot43(e1, pVec);
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (det > -kEpsilon && det < kEpsilon) return false;
#endif
  float invDet = 1.0f / det;
  float4 tVec =
      make_float4(origin.x - a.x, origin.y - a.y, origin.z - a.z, 0.0f);
  float4 qVec =
      make_float4(tVec.y * e1.z - tVec.z * e1.y, tVec.z * e1.x - tVec.x * e1.z,
                  tVec.x * e1.y - tVec.y * e1.x, 0.0f);
  float t = e2.x * qVec.x;
  t += e2.y * qVec.y;
  t += e2.z * qVec.z;
  t *= invDet;
// Do not allow ray origin in front of triangle
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (t < 0.0f) return false;
#endif
  float u = tVec.x * pVec.x;
  u += tVec.y * pVec.y;
  u += tVec.z * pVec.z;
  u *= invDet;
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (u < 0.0f || u > 1.0f) return false;
#endif
  float v = dir.x * qVec.x;
  v += dir.y * qVec.y;
  v += dir.z * qVec.z;
  v *= invDet;
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (v < 0.0f || u + v > 1.0f) return false;
#endif
  isect.t = t;
  isect.triId = triId;
  isect.u = u;
  isect.v = v;
#ifdef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  return t >= 0.0f & u >= 0.0f & u <= 1.0f & v >= 0.0f & ((u + v) <= 1.0f);
#else
  return true;
#endif
}

inline __host__ __device__ __host__ void createEvents0(
    const float4& origin, const float4& direction, const float4& invDirection,
    const float4& center, const float4& hit, float tNear, float tFar,
    OctreeEvent* events, int16_t* N) {
  float4 diff_center_origin = make_float4(
      center.x - origin.x, center.y - origin.y, center.z - origin.z, 0.0f);
  float4 t = make_float4(diff_center_origin.x * invDirection.x,
                         diff_center_origin.y * invDirection.y,
                         diff_center_origin.z * invDirection.z, 0.0f);
  // Create the events, unsorted.
  events[1].type = OCTREE_EVENT_X;
  events[1].mask = 0x1;
  events[1].t = t.x;
  events[2].type = OCTREE_EVENT_Y;
  events[2].mask = 0x2;
  events[2].t = t.y;
  events[3].type = OCTREE_EVENT_Z;
  events[3].mask = 0x4;
  events[3].t = t.z;
  // Sort the planarEvents, so we can implement a front-to-back traversal.
  exchangeIf(
      !isValidT(events[2].t, tNear, tFar) |
          (events[2].t > events[3].t & isValidT(events[3].t, tNear, tFar)),
      &events[0], &events[2], &events[3]);
  exchangeIf(
      !isValidT(events[1].t, tNear, tFar) |
          (events[1].t > events[2].t & isValidT(events[2].t, tNear, tFar)),
      &events[0], &events[1], &events[2]);
  exchangeIf(
      !isValidT(events[2].t, tNear, tFar) |
          (events[2].t > events[3].t & isValidT(events[3].t, tNear, tFar)),
      &events[0], &events[2], &events[3]);
  // Discard planarEvents with t > tFar.
  // k is the index of the last event.
  int k = 2;
  while (k >= 0 && !isValidT(events[k + 1].t, tNear, tFar)) --k;
  // Consolidate planarEvents that have the same t-value.
  // There are only 1, 2, or 3 planarEvents, so we just explicitly compute
  // this.
  if (k == 2) {
    bool left_equal = (events[1].t == events[2].t);
    bool right_equal = (events[2].t == events[3].t);
    if (left_equal && right_equal) {
      events[1].mask = events[1].mask | events[2].mask | events[3].mask;
      k = 0;
    } else if (left_equal) {
      events[1].mask = events[1].mask | events[2].mask;
      events[2] = events[3];
      k = 1;
    } else if (right_equal) {
      events[2].mask = events[2].mask | events[3].mask;
      k = 1;
    }
  } else if (k == 1) {
    if (events[1].t == events[2].t) {
      events[1].mask = events[1].mask | events[2].mask;
      k = 0;
    }
  }
  unsigned char xBit = (hit.x > center.x);
  unsigned char yBit = (hit.y > center.y);
  unsigned char zBit = (hit.z > center.z);
  events[0].type = OCTREE_EVENT_ENTRY;
  events[0].t = tNear;
  events[0].mask = xBit | (yBit << 1) | (zBit << 2);
  events[k + 2].type = OCTREE_EVENT_EXIT;
  events[k + 2].t = tFar;
  events[k + 2].mask = 0;
  *N = (k + 1) + 2;
  unsigned char xMask =
      (events[1].type == OCTREE_EVENT_X) & ((xBit == 1) | (direction.x < 0.0f));
  unsigned char yMask =
      (events[1].type == OCTREE_EVENT_Y) & ((yBit == 1) | (direction.y < 0.0f));
  unsigned char zMask =
      (events[1].type == OCTREE_EVENT_Z) & ((zBit == 1) | (direction.z < 0.0f));
  unsigned char mask = xMask | (yMask << 1) | (zMask << 2);
  //  if ((k + 1) + 2 > 2 && events[0].t == events[1].t)
  events[0].mask =
      events[0].mask ^ (((k + 1) + 2 > 2 && events[0].t == events[1].t) * mask);
}

inline float4 __device__ __host__ getSplitPoint(const OctNode128* node,
                                                const Aabb4& bounds) {
  const float4& min = bounds.min;
  const float4& max = bounds.max;
  uint16_t num_samples = node->footer.internal.sizeDescriptor;
  if (num_samples <= 1) return 0.5f * (min + max);
  float inv = 1.0 / (num_samples - 1);
  float4 step_size = inv * (max - min);
  float4 split_point =
      make_float4(node->footer.internal.i, node->footer.internal.j,
                  node->footer.internal.k, 0.0f);
  split_point *= step_size;
  split_point += min;
  return split_point;
}

/*#define DEBUG_TRAVERSE*/
#ifdef DEBUG_TRAVERSE
//#define DEBUG_TRAVERSE_THREAD_ID 389308
#define DEBUG_TRAVERSE_THREAD_ID 124009
//#define DEBUG_TRAVERSE_THREAD_ID 386858  // t = 0.485482
#endif
#define MAX_DEPTH 15
#define MAX_EVENTS 4
#define STACK_SIZE (MAX_EVENTS * MAX_DEPTH)
#ifdef USE_PERSISTENT

#endif
inline __device__ void intersectOctree(
    const float4* rays, const uint4* nodes, const float4* vertices,
    const int4* indices, const uint32_t* references, const Aabb4 bounds,
    uint32_t numTriangles, uint32_t numVertices, uint32_t numRays, Hit* hits,
    int width, int height, size_t hitPitch, size_t rayPitch) {
  // NOTE:
  //    1) We need to examine 4 nodes per octree node.
  //    4) Because of (1), we create a stack of size:
  //          4 * d  * B
  //    where we need B bytes per node to store a reference on the stack.
  // Here B = 4, since unsigned ints will be used.
  //
  // NOTE: With treelet demarcations, we could allow treelets
  // of maximum size 16k nodes, so short ints could be use where B_16 = 2.
  //
  // NOTE: This uses thread-local storage - it is really global memory as
  // opposed to shared memory.  The danger of using shared memory is that
  // many threads may fetch the same node, so it might be best to let
  // the GPU manage the cache on its own and hopefully we only fetch
  // each node that we actually need.
  const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t warpId = tid / WARP_SIZE;       // get our warpId
  const unsigned char laneId = tid % WARP_SIZE;  // get our warp index
  const uint32_t warpIdx = warpId % WARPS_PER_BLOCK;
  int nodeIdStack[STACK_SIZE];
  Aabb4 aabbStack[STACK_SIZE];
  float tNearStack[STACK_SIZE];
  float tFarStack[STACK_SIZE];
  __shared__ volatile int localRayCount[WARPS_PER_BLOCK];
  __shared__ volatile int localNextRay[WARPS_PER_BLOCK];

  localNextRay[warpIdx] = 0;
  localRayCount[warpIdx] = 0;

  do {
    // If we are the first thread in the warp, check our work status
    // and add more work if needed.
    if (laneId == 0 && localRayCount[warpIdx] <= 0) {
      localNextRay[warpIdx] = atomicAdd(&nextRayIndex, WARP_BATCH_SIZE);
      localRayCount[warpIdx] = WARP_BATCH_SIZE;
    }

    // Get the next ray for this thread.
    int rayIdx = localNextRay[warpIdx] + laneId;
#if 0
    int x = rayIdx % width;
    int y = rayIdx / width;
    if (x == 300 && y == 300) {
      const float4 o = GET_RAY_ORIGIN(rays, width, rayPitch, rayIdx);
      const float4 d = GET_RAY_DIRECTION(rays, width, rayPitch, rayIdx);
      uint32_t row_offset = (rayIdx / width) * rayPitch;
      uint32_t col_offset = 2 * (rayIdx % width);
      const float4* pos =
          reinterpret_cast<const float4*>(reinterpret_cast<const char*>(rays) +
                                          row_offset) +
          col_offset;
      printf(
          "[%d] x = %d y = %d width = %d, pitch = %ld, pos = %lx, o = %f %f "
          "%f, "
          "d "
          "= %f %f "
          "%f\n",
          tid, x, y, width, rayPitch, pos, o.x, o.y, o.z, d.x, d.y, d.z);
    }
#endif

#ifdef DEBUG_TRAVERSE
    int numNodes = 0;
    int numLeaves = 0;
    int depthStack[STACK_SIZE];
    int depth = 0;
    depthStack[0] = 0;
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      const float4 o = GET_RAY_ORIGIN(rays, width, rayPitch, rayIdx);
      const float4 d = GET_RAY_DIRECTION(rays, width, rayPitch, rayIdx);
      printf("[%d] o = %f %f %f, d = %f %f %f\n", tid, o.x, o.y, o.z, d.x, d.y,
             d.z);
    }
#endif
    bool goodThread = rayIdx < numRays;
    if (!goodThread) break;

    // Update counts and next ray to get.
    if (laneId == 0) {
      localNextRay[warpIdx] += WARP_SIZE;
      localRayCount[warpIdx] -= WARP_SIZE;
    }

    // Initialize traversal.
    const float4 origin = GET_RAY_ORIGIN(rays, width, rayPitch, rayIdx);
    const float4 dir = GET_RAY_DIRECTION(rays, width, rayPitch, rayIdx);
    const float4 invDirection =
        make_float4(1.0f / dir.x, 1.0f / dir.y, 1.0f / dir.z, 0.0f);
    int16_t stackEnd = 1;
    int currentId = -1;
    float tNear = 0.0f, tFar = 0.0f;
    bool stackEmpty = false;
    bool objectHit = false;

    Hit closest;
    closest.t = NPP_MAXABS_32F;
    closest.triId = -1;

    // Put the root onto the stack.
    nodeIdStack[0] = 0;
    aabbStack[0] = bounds;
    bool hitBounds =
        intersectAabb2(origin, invDirection, aabbStack[0], 0.0f, NPP_MAXABS_32F,
                       &tNearStack[0], &tFarStack[0]);
#ifdef DEBUG_TRAVERSE
    int x = rayIdx % width;
    int y = rayIdx / width;
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      printf(
          "x = %d, y = %d, hitBounds = %d, objectHit = %d, stackEmpty = %d "
          "tNear = % f tFar "
          "= "
          "% f\n",
          x, y, hitBounds, objectHit, stackEmpty, tNearStack[0], tFarStack[0]);
    }
#endif

    while (hitBounds & !(objectHit | stackEmpty)) {
      // Setup beore entering loop.
      stackEmpty = (stackEnd <= 0);
      currentId = nodeIdStack[!stackEmpty * (stackEnd - 1)];
      OctNode128 currentNode;
      if (!stackEmpty) {
        *reinterpret_cast<uint4*>(&currentNode) =
            tex1Dfetch(texture_nodes, currentId);
        /*currentNode = *reinterpret_cast<const
         * OctNode128*>(&nodes[currentId]);*/
      }
      bool foundLeaf = (currentNode.header.type == NODE_LEAF) && !stackEmpty;
      tNear = !stackEmpty * tNearStack[!stackEmpty * (stackEnd - 1)];
      tFar = !stackEmpty * tFarStack[!stackEmpty * (stackEnd - 1)];

      // Go until stack empty or found a leaf.
      while (!foundLeaf && !stackEmpty) {
        // Get node information.
        currentId = nodeIdStack[stackEnd - 1];
        Aabb4 currentBounds = aabbStack[stackEnd - 1];
#ifdef DEBUG_TRAVERSE
        ++numNodes;
        depth = depthStack[stackEnd - 1];
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          printf("[N $%x #%d @%d +%d %d, %d, %d, %d] %f %f\n",
                 currentNode.footer.internal.childMask,
                 countBits(currentNode.footer.internal.childMask),
                 currentNode.header.octant, currentNode.header.offset,
                 currentNode.footer.internal.i, currentNode.footer.internal.j,
                 currentNode.footer.internal.k,
                 currentNode.footer.internal.sizeDescriptor, tNear, tFar);
        }
#endif
        float4 hit =
            make_float4(origin.x + tNear * dir.x, origin.y + tNear * dir.y,
                        origin.z + tNear * dir.z, 0.0f);
        float4 center = getSplitPoint(&currentNode, currentBounds);

        //  Get the events, in order of they are hit.
        int16_t numEvents = 0;
        OctreeEvent events[5];
        int16_t numValidEvents = 0;
        createEvents0(origin, dir, invDirection, center, hit, tNear, tFar,
                      events, &numEvents);
        unsigned char octantBits = 0x0;

        // Get children.
        uint32_t children[8];
        uint32_t childId = currentNode.header.offset;
        octantBits = currentNode.footer.internal.childMask;
#pragma unroll
        for (uint32_t i = 0; i < 8; ++i) {
          children[i] = childId;
          childId += ((octantBits >> i) & 0x1);
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {*/
/*printf("%d ", children[i]);*/
/*}*/
#endif
        }
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {*/
/*printf("\n");*/
/*}*/
#endif
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          int numChildren = countBits(octantBits);
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          for (int i = 0; i < numChildren; ++i) {
            OctNode128 child = *reinterpret_cast<const OctNode128*>(
                &nodes[currentNode.header.offset + i]);
            if (child.header.type == NODE_LEAF) {
              printf("L %d %d %d, ", child.header.octant, child.header.offset,
                     child.footer.leaf.size);
            } else {
              printf("N %d, ", child.header.octant);
            }
          }
          printf("\n");
        }
#endif

        // Figure which octants were hit are non-empty.
        unsigned char octant = 0x0;
        numValidEvents = 0;
        for (int16_t i = 0; i < numEvents - 1; ++i) {
          octant = octant ^ events[i].mask;
          bool hasChild = ((octantBits & (0x1 << octant)) != 0);
          numValidEvents += hasChild;
        }
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          octant = 0x0;
          printf("[%08d] #%d", currentId, numEvents);
          for (int i = 0; i < depth; ++i) printf("  ");
          for (int i = 0; i < numEvents - 1; ++i) {
            octant = octant ^ events[i].mask;
            printf("(%d, %f, %x, %d) ", octant, events[i].t, events[i].mask,
                   ((octantBits & (0x1 << octant)) != 0));
          }
          printf("\n");
        }
#endif

#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("add -->");
        }
#endif
        // Add the children in reverse order of being hit to the stack.  This
        // way,  the child that was hit first gets popped first.
        int16_t k = -1;  // keep track of which valid event we have
        octant = 0x0;
        for (int16_t i = 0; (i < numEvents - 1) & ((k + 1) < numValidEvents);
             ++i) {
          octant = octant ^ events[i].mask;
          bool hasChild = ((octantBits & (0x1 << octant)) != 0);
          k += hasChild;
          int16_t nextStack = (stackEnd - 1) + numValidEvents - k - 1;
          if (hasChild) {  // divergence
#ifdef DEBUG_TRAVERSE
            if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
              printf("%d ", children[octant]);
            }
#endif
            nodeIdStack[nextStack] = children[octant];
            aabbStack[nextStack] =
                getChildBounds(currentBounds, center, octant);
            tNearStack[nextStack] = events[i].t;
            tFarStack[nextStack] = events[i + 1].t;
#ifdef DEBUG_TRAVERSE
            depthStack[nextStack] = depth + 1;
#endif
          }
        }
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("\n");
        }
#endif
        stackEnd += numValidEvents;
        --stackEnd;
        stackEmpty = (stackEnd <= 0);
        currentId = nodeIdStack[!stackEmpty * (stackEnd - 1)];
        if (!stackEmpty) {
          *reinterpret_cast<uint4*>(&currentNode) =
              tex1Dfetch(texture_nodes, currentId);
          /*currentNode = *reinterpret_cast<const
           * OctNode128*>(&nodes[currentId]);*/
        }
        foundLeaf = (currentNode.header.type == NODE_LEAF) && !stackEmpty;
        tNear = !stackEmpty * tNearStack[!stackEmpty * (stackEnd - 1)];
        tFar = !stackEmpty * tFarStack[!stackEmpty * (stackEnd - 1)];
      }  // end of while (!foundLeaf && !stackEmpty)
#ifdef DEBUG_TRAVERSE
      if (foundLeaf) {
        ++numLeaves;
        depth = depthStack[stackEnd - 1];
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          uint32_t size = currentNode.footer.leaf.size;
          uint32_t offset = currentNode.header.offset;
          uint32_t octant = currentNode.header.octant;
          printf("[L #%d @%d +%d] %f %f\n", size, octant, offset, tNear, tFar);
        }
      }
#endif

      // We either have a leaf or stack is empty.
      uint32_t numPrimitives = currentNode.footer.leaf.size;
      uint32_t offset = currentNode.header.offset;
      bool triangleHit = false;
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {*/
/*printf("-->[L #%d @%d +%d]\n", numPrimitives, octant, offset);*/
/*}*/
#endif

#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("[%08d]", currentId);
        for (int i = 0; i < depth; ++i) printf("  ");
      }
#endif
      numPrimitives *= !stackEmpty;
      offset *= !stackEmpty;
      for (uint32_t i = 0; i < numPrimitives; ++i) {
        uint32_t triId = references[i + offset];
        /*uint32_t triId = tex1Dfetch(texture_references, i + offset);*/
        Hit isect;
        isect.t = NPP_MAXABS_32F;
        isect.triId = -1;
        bool isNewClosest =
            intersectTriangle(origin, dir, indices, vertices, triId, isect,
                              numTriangles, numVertices) &&
            isect.t >= tNear && isect.t <= tFar && isect.t < closest.t;
        if (isNewClosest) closest = isect;
        /*closest.t = isNewClosest * isect.t + !isNewClosest * closest.t;*/
        /*closest.triId =*/
        /*isNewClosest * isect.triId + !isNewClosest * closest.triId;*/
        /*closest.u = isNewClosest * isect.u + !isNewClosest * closest.u;*/
        /*closest.v = isNewClosest * isect.v + !isNewClosest * closest.v;*/
        triangleHit = isNewClosest || triangleHit;
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
          printf("(%f, %d)", isect.t, isNewClosest);
        }
#endif
      }
#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("\n");
      }
#endif
      objectHit = triangleHit && (closest.t >= tNear) && (closest.t <= tFar);
      if (foundLeaf) --stackEnd;
#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("[%08d]", currentId);
        for (int i = 0; i < depth; ++i) printf("  ");
        printf("hit = %d numPrimitives = %d offset = %d t_best = %f\n",
               objectHit, numPrimitives, offset, closest.t);
      }
#endif
    }  // end of while (!(objectHit | stackEmpty))

#ifdef DEBUG_TRAVERSE
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      printf("numNodes = %d numLeaves = %d\n", numNodes, numLeaves);
    }
#endif
#if 0 
    if (x == 300 && y == 300) {
      printf("hitPitch = %ld\n", hitPitch);
    }
#endif
    SET_HIT(hits, width, hitPitch, rayIdx, closest);
#ifdef DEBUG_TRAVERSE
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      printf("[%d] t=%f triId=%d u=%f v=%f hit=%d\n", rayIdx, closest.t,
             closest.triId, closest.u, closest.v, objectHit);
    }
#endif
  } while (true);
}

__global__
#ifdef USE_TRACE_KERNEL_LAUNCH_BOUNDS
    __launch_bounds__(THREADS_PER_BLOCK, MIN_BLOCKS)
#endif
        void traceKernel(const __restrict__ float4* rays,
                         const __restrict__ uint4* nodes,
                         const __restrict__ float4* vertices,
                         const __restrict__ int4* indices,
                         const __restrict__ uint32_t* references,
                         const Aabb4 bounds, uint32_t numTriangles,
                         uint32_t numVertices, uint32_t numRays, Hit* hits,
                         int width, int height, size_t hitPitch,
                         size_t rayPitch) {
  intersectOctree(rays, nodes, vertices, indices, references, bounds,
                  numTriangles, numVertices, numRays, hits, width, height,
                  hitPitch, rayPitch);
}

CUDAOctreeRenderer::CUDAOctreeRenderer(const ConfigLoader& c) : config(c) {
  image.filename = config.imageFilename;
  image.width = config.imageWidth;
}

CUDAOctreeRenderer::CUDAOctreeRenderer(const ConfigLoader& c,
                                       const BuildOptions& options)
    : config(c), buildOptions(options) {
  image.filename = config.imageFilename;
  image.width = config.imageWidth;
}

__global__ void generateRaysKernel(uint32_t width, uint32_t height, float fov,
                                   float focal_distance, float3 eye,
                                   float3 tangent, float3 up, float3 look,
                                   float4* d_rays, size_t pitch) {
  const uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  const uint32_t warpId = tid / WARP_SIZE;       // get our warpId
  const unsigned char laneId = tid % WARP_SIZE;  // get our warp index
  const uint32_t warpIdx = warpId % WARPS_PER_BLOCK;
  const uint32_t numRays = width * height;
  __shared__ volatile int localRayCount[WARPS_PER_BLOCK];
  __shared__ volatile int localNextRay[WARPS_PER_BLOCK];

  localNextRay[warpIdx] = 0;
  localRayCount[warpIdx] = 0;

  do {
    // If we are the first thread in the warp, check our work status
    // and add more work if needed.
    if (laneId == 0 && localRayCount[warpIdx] <= 0) {
      localNextRay[warpIdx] = atomicAdd(&nextRayIndex, WARP_BATCH_SIZE);
      localRayCount[warpIdx] = WARP_BATCH_SIZE;
    }

    // Get the next ray for this thread.
    int rayIdx = localNextRay[warpIdx] + laneId;

    bool goodThread = rayIdx < numRays;
    if (!goodThread) break;

    // Update counts and next ray to get.
    if (laneId == 0) {
      localNextRay[warpIdx] += WARP_SIZE;
      localRayCount[warpIdx] -= WARP_SIZE;
    }

    // Compute parameters needed to get the ray direction.
    // This is done in eye coordinates.
    int y = rayIdx / width;  // Get the y coordinate in screen space.
    float y_max = focal_distance *
                  tan(((M_PI / 180.0f) * fov) / 2.0f);  // Find maxiumum eye Y.
    float v = (2.0f * y) / height - 1.0f;
    float eye_y = v * y_max;  // Get eye space Y.
    int x = rayIdx % width;   // Get x coordinate in screen space.
    float aspect = (1.0f * width) / height;
    float x_max = aspect * y_max;
    float u = (2.0f * x) / width - 1.0f;
    float eye_x = u * x_max;  // Get the eye space X.

    // Compute and set the origin and direction here.
    float4* pos =
        reinterpret_cast<float4*>(reinterpret_cast<char*>(d_rays) + pitch * y) +
        2 * x;  // Get the location to the values we are going to set.
    float3 origin = eye - focal_distance * look + eye_x * tangent + eye_y * up;
    *pos = make_float4(origin, 0.0f);  // Set the origin.
    float3 direction = normalize(origin - eye);
    *(pos + 1) = make_float4(direction, NPP_MAXABS_32F);  // Set the direction.
  } while (true);
}

void CUDAOctreeRenderer::sortRays(uint32_t width, bool usePitched, int numRays,
                                  size_t* rankInOutPitch, float4* d_rays,
                                  int2** d_rank_in_out) {
  // Get the allocation out of the way.
  *rankInOutPitch = 2 * sizeof(int2) * width;
  if (usePitched)
    CHK_CUDA(hipMallocPitch(d_rank_in_out, rankInOutPitch,
                             width * sizeof(int2) * 2, height))
  else
    CHK_CUDA(hipMalloc(d_rank_in_out, sizeof(int2) * 2 * width * height));
}

void CUDAOctreeRenderer::generateRays(uint32_t width, uint32_t height,
                                      float focal_distance, float fov,
                                      const float3& eye, const float3& center,
                                      const float3& up, bool sort,
                                      bool usePitched, float4** d_rays,
                                      int* numRays, size_t* pitch) {
  image.width = width;
  image.height = height;

  *pitch = 2 * sizeof(float4) * width;

  if (usePitched)
    CHK_CUDA(hipMallocPitch(d_rays, pitch, width * sizeof(float4) * 2, height))
  else
    CHK_CUDA(hipMalloc(d_rays, sizeof(float4) * 2 * width * height));

  *numRays = width * height;

  // Set memory configuration.
  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  // Compute warps and blocks.
  const int numWarps = 32 * 5;
  const uint32_t numThreadsPerBlock = THREADS_PER_BLOCK;
  const uint32_t numBlocks = (numWarps + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;

  // Initialize global state.
  uint32_t nextRay = 0;
  CHK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(nextRayIndex), &nextRay, sizeof(uint32_t)));

  // Intialize some local state.

  // Compute left handed coordinate system camera orientation.
  float3 z_axis = normalize(eye - center);  // We look down negative Z.
  float3 x_axis = normalize(cross(normalize(up), z_axis));  // Get tangent.
  float3 y_axis = normalize(cross(z_axis, x_axis));  // True up direction.
  printf("tangent = %f %f %f up = %f %f %f look = %f %f %f\n", x_axis.x,
         x_axis.y, x_axis.z, y_axis.x, y_axis.y, y_axis.z, z_axis.x, z_axis.y,
         z_axis.z);

  // Call our kernel.
  generateRaysKernel<<<numBlocks, numThreadsPerBlock>>>(
      width, height, fov, focal_distance, eye, x_axis, y_axis, z_axis, *d_rays,
      *pitch);

  CHK_CUDA(hipDeviceSynchronize());
}

void CUDAOctreeRenderer::loadScene() {
  SceneLoader loader(config.objFilename);
  loader.load(&scene);
}

void CUDAOctreeRenderer::render() {
  int4* d_indices;
  float4* d_vertices;

  // Clear the device.  It is ours now.
  CHK_CUDA(hipDeviceReset());

  loadScene();

  CHK_CUDA(hipMalloc((void**)&d_indices, scene.numTriangles * sizeof(int4)));
  CHK_CUDA(
      hipMalloc((void**)&d_vertices, scene.numTriangles * sizeof(float4)));

  LOG(DEBUG) << "numTriangles = " << scene.numTriangles << " "
             << " numVertices = " << scene.numVertices << "\n";

  int4* indices = scene.indices;

  CHK_CUDA(hipMemcpy(d_indices, indices, scene.numTriangles * sizeof(int4),
                      hipMemcpyHostToDevice));

  float4* vertices = scene.vertices;

  CHK_CUDA(hipMemcpy(d_vertices, vertices, scene.numVertices * sizeof(float4),
                      hipMemcpyHostToDevice));

  traceOnDevice(d_indices, d_vertices);

  CHK_CUDA(hipFree(d_indices));
  CHK_CUDA(hipFree(d_vertices));

  CHK_CUDA(hipDeviceReset());
}

void CUDAOctreeRenderer::buildOnDevice(Octree<LAYOUT_AOS>* d_octree) {}

#define DEBUG_CHECK_FILE_OCTREE
void CUDAOctreeRenderer::buildFromFile(Octree<LAYOUT_AOS>* d_octree) {
  Octree<LAYOUT_AOS> octreeFileAos;
  LOG(DEBUG) << "Building from: " << buildOptions.info << "\n";
  octreeFileAos.buildFromFile(buildOptions.info);
  octreeFileAos.setGeometry(NULL, NULL, scene.numTriangles, scene.numVertices);
  Octree<LAYOUT_AOS> octreeFileSoa;
  octreeFileSoa.copy(octreeFileAos);
  octreeFileSoa.copyToGpu(d_octree);
#ifdef DEBUG_CHECK_FILE_OCTREE
  Octree<LAYOUT_AOS> octreeFileSoaCheck;
  octreeFileSoaCheck.copyFromGpu(d_octree);
  LOG(DEBUG) << octreeFileSoaCheck << "\n";
#endif
}

void CUDAOctreeRenderer::build(Octree<LAYOUT_AOS>* d_octree) {
  switch (buildOptions.type) {
    case BuildOptions::BUILD_FROM_FILE:
      buildFromFile(d_octree);
      break;
    case BuildOptions::BUILD_ON_DEVICE:
      buildOnDevice(d_octree);
      break;
    default:
      break;
  }
}

void CUDAOctreeRenderer::traceOnDevice(int4* indices, float4* vertices) {
  const int numThreadsPerBlock = THREADS_PER_BLOCK;
  const int numWarps = 32 * 5;

  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  //      (numRays + WARP_BATCH_SIZE - 1) / WARP_BATCH_SIZE;
  const int numBlocks = (numWarps + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;

  // Allocate rays.
  /*CHK_CUDA(hipMalloc(&d_rays, sizeof(Ray) * numRays));*/

  // Generate rays.
  size_t rayPitch = image.width * sizeof(float4) * 2;
  bool usePitched = true;
  bool sort = false;
  image.width = config.imageWidth;
  image.height = config.imageHeight;
  generateRays(image.width, image.height, config.focal_distance, config.fov,
               config.eye, config.center, config.up, sort, usePitched,
               reinterpret_cast<float4**>(&d_rays), &numRays, &rayPitch);

  // Allocate hits for results.
  size_t hitPitch = image.width * sizeof(Hit);
  if (usePitched)
    CHK_CUDA(hipMallocPitch(&d_hits, &hitPitch, sizeof(Hit) * image.width,
                             image.height))
  else
    CHK_CUDA(hipMalloc(&d_hits, sizeof(Ray) * image.width * image.height));

  std::vector<Hit> initialHits(numRays);
  const Hit badHit = {0.0f, -1, 0.0f, 0.0f};

  // Initialize to non-hit.
  for (int i = 0; i < numRays; ++i) initialHits[i] = badHit;
  CHK_CUDA(hipMemcpy2D(d_hits, hitPitch, &initialHits[0],
                        sizeof(Hit) * image.width, sizeof(Hit) * image.width,
                        image.height, hipMemcpyHostToDevice));

  LOG(DEBUG) << "WARP_LOAD_FACTOR = " << WARP_LOAD_FACTOR
             << " WARPS_PER_BLOCK = " << WARPS_PER_BLOCK
             << " numRays = " << numRays << " numWarps = " << numWarps
             << " numThreadsPerBlock = " << numThreadsPerBlock
             << " numBlocks = " << numBlocks << " hitPitch = " << hitPitch
             << " rayPitch = " << rayPitch
             << " numThreads = " << numBlocks * THREADS_PER_BLOCK << "\n";

  Octree<LAYOUT_AOS>* d_octree = NULL;
  CHK_CUDA(hipMalloc((void**)(&d_octree), sizeof(Octree<LAYOUT_AOS>)));

  build(d_octree);

#ifdef UPDATE_HITS_SOA
  LOG(DEBUG) << "Using SOA format for hits.\n";
#endif
  LOG(DEBUG) << "Ray tracing...\n";

  // OK, let's bind textures.
  const NodeStorage<LAYOUT_AOS>* d_nodeStorage = d_octree->nodeStoragePtr();
  uint32_t numNodes = 0;
  CHK_CUDA(hipMemcpy(&numNodes, &(d_nodeStorage->numNodes), sizeof(uint32_t),
                      hipMemcpyDeviceToHost));
  LOG(DEBUG) << "numNodes = " << numNodes << "\n";
  uint4* d_nodes;
  CHK_CUDA(hipMemcpy(&d_nodes, &(d_nodeStorage->nodes), sizeof(OctNode128*),
                      hipMemcpyDeviceToHost))
  hipBindTexture(0, texture_nodes, d_nodes, sizeof(OctNode128) * numNodes);
  hipBindTexture(0, texture_vertices, vertices,
                  scene.numVertices * sizeof(float4));
  hipBindTexture(0, texture_indices, indices,
                  scene.numTriangles * sizeof(int4));
  uint32_t numReferences = 0;
  CHK_CUDA(hipMemcpy(&numReferences, d_octree->numTriangleReferencesPtr(),
                      sizeof(uint32_t), hipMemcpyDeviceToHost));
  LOG(DEBUG) << "numReferences = " << numReferences << "\n";
  uint32_t* d_references;
  CHK_CUDA(hipMemcpy(&d_references, d_octree->triangleIndicesPtr(),
                      sizeof(uint32_t*), hipMemcpyDeviceToHost));
  hipBindTexture(0, texture_references, d_references,
                  numReferences * sizeof(int));

  uint32_t nextRay = 0;
  CHK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(nextRayIndex), &nextRay, sizeof(uint32_t)));
  Aabb4 bounds;
  bounds.min = make_float4(scene.bbmin, 0.0);
  bounds.max = make_float4(scene.bbmax, 0.0);
  float time = 0.0f;
  float avg_time = 0.0f;
  hipEvent_t start_event, stop_event;
#if 0 
  size_t logLimit = 0;
  hipDeviceGetLimit(&logLimit, hipLimitPrintfFifoSize);
  printf("--->Old logLimit = %d\n", logLimit);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 10 * logLimit);
  hipDeviceGetLimit(&logLimit, hipLimitPrintfFifoSize);
  printf("--->New logLimit = %d\n", logLimit);
#endif
  int warmup_trials = 10;
  int run_trials = 10;
  int total_trials = warmup_trials + run_trials;
  for (int i = 0; i < total_trials; ++i) {
    nextRay = 0;
    CHK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(nextRayIndex), &nextRay, sizeof(uint32_t)));
    CHK_CUDA(hipEventCreate(&start_event));
    CHK_CUDA(hipEventCreate(&stop_event));
    CHK_CUDA(hipEventRecord(start_event, 0));
    traceKernel<<<numBlocks, numThreadsPerBlock>>>(
        reinterpret_cast<float4*>(d_rays), d_nodes, vertices, indices,
        d_references, bounds, scene.numTriangles, scene.numVertices, numRays,
        d_hits, image.width, image.height, hitPitch, rayPitch);
    CHK_CUDA(hipEventRecord(stop_event, 0));
    CHK_CUDA(hipEventSynchronize(stop_event));
    hipEventElapsedTime(&time, start_event, stop_event);
    if (i >= warmup_trials) avg_time += time;
  }
  avg_time /= run_trials;

  LOG(DEBUG) << "Done...\n";
  LOG(DEBUG) << "Average time = " << avg_time * 1000.0 << " microsec"
             << ", " << avg_time << " millisec\n";
  float ns_per_ray = 1000000.0 * avg_time / numRays;
  LOG(DEBUG) << "Traced " << numRays << " rays at " << ns_per_ray
             << " nanoseconds per ray\n";
  float mrays_sec = numRays / (1000.0 * avg_time);
  LOG(DEBUG) << "Rate is " << mrays_sec << " million rays per second.\n";
  Octree<LAYOUT_AOS>::freeOnGpu(d_octree);
  CHK_CUDA(hipFree((void*)(d_octree)));
#ifdef UPDATE_HITS_SOA
  LOG(DEBUG) << "Converting hits from SOA to AOS.\n";
  reorderHitsKernel<<<numBlocks, numThreadsPerBlock>>>(d_hits, numRays);
  hipDeviceSynchronize();
  LOG(DEBUG) << "SOA to AOS conversion done.\n";
#endif

  // Copy hits locally.
  localHits.resize(numRays);
  if (usePitched)
    CHK_CUDA(hipMemcpy2D(&localHits[0], hitPitch, d_hits,
                          sizeof(Hit) * image.width, sizeof(Hit) * image.width,
                          image.height, hipMemcpyDeviceToHost))
  else
    CHK_CUDA(hipMemcpy(&localHits[0], d_hits, sizeof(Hit) * numRays,
                        hipMemcpyDeviceToHost));

  CHK_CUDA(hipFree(d_hits));
  CHK_CUDA(hipFree(d_rays));
}

void CUDAOctreeRenderer::shade() {
  image.resize();

  float3 backgroundColor = {0.2f, 0.2f, 0.2f};

  Hit* hits = &localHits[0];

  for (size_t i = 0; i < numRays; i++) {
    if (hits[i].triId < 0) {
      image.pixel[i] = backgroundColor;
#if 0
      int width = image.width;
      int x = i % width;
      int y = i / width;
      if ((x > 100 && x < 106) && (y > 200 && y < 250)) {
        image.pixel[i] = make_float3(1.0f, 0.0f, 0.0f);
        /*std::cout << "i = " << i << " x = " << x << " y = " << y << "\n";*/
      }
#endif
    } else {
      if (hits[i].triId > scene.numTriangles) {
#if 0
        std::cout << " Got out of bounds triangle ID: " << hits[i].triId << "\n";
#endif
        continue;
      }
      const int4 tri = scene.indices[hits[i].triId];
      const float4 v0 = scene.vertices[tri.x];
      const float4 v1 = scene.vertices[tri.y];
      const float4 v2 = scene.vertices[tri.z];
      const float4 e0 = v1 - v0;
      const float4 e1 = v2 - v0;
      const float3 n = normalize(
          cross(make_float3(e0.x, e0.y, e0.z), make_float3(e1.x, e1.y, e1.z)));

      image.pixel[i] = 0.5f * n + make_float3(0.5f, 0.5f, 0.5f);
    }
  }
}

}  // namespace oct
