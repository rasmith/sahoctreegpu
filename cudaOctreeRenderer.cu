#include "hip/hip_runtime.h"
#include "cudaOctreeRenderer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nppdefs.h>
#include <float.h>
#include <sys/time.h>

#include "log.h"
#include "octree.h"
#include "cuda_math.h"

#define kEpsilon 1e-18

#define USE_PERSISTENT

#define WARP_SIZE 32   // Hardware size of a warp, 32 lanes.
#define WARP_FACTOR 4  // How many warps per block do we want.
#define THREADS_PER_BLOCK (WARP_FACTOR * WARP_SIZE)  // Compute # threads.
#define WARPS_PER_BLOCK WARP_FACTOR
#define REGISTERS_PER_SM (1 << 15)
#define SHARED_MEMORY_PER_SM (1 << 15)
#define MAX_REGISTERS_THREAD 63
#define MIN_BLOCKS \
  ((REGISTERS_PER_SM) / (THREADS_PER_BLOCK * MAX_REGISTERS_THREAD))
#define MAX_SHARED_MEMORY_PER_BLOCK SHARED_MEMORY_PER_SM / MIN_BLOCKS
#define MAX_BLOCKS_PER_DIMENSION 65535
//#define UPDATE_HITS_SOA

#define WARP_LOAD_FACTOR 2  // This is effectively #rays / threads
#define WARP_BATCH_SIZE (WARP_LOAD_FACTOR * WARP_SIZE)  // #rays / warp batch
__device__ int nextRayIndex;

//#define USE_TRACE_KERNEL_LAUNCH_BOUNDS
texture<float4, 1, hipReadModeElementType> texture_rays;
texture<uint4, 1, hipReadModeElementType> texture_nodes;
texture<float4, 1, hipReadModeElementType> texture_vertices;
texture<int4, 1, hipReadModeElementType> texture_indices;
texture<uint32_t, 1, hipReadModeElementType> texture_references;

namespace oct {

template <uint32_t N>
__host__ __device__ inline uint32_t lg2() {
  return ((N >> 1) != 0) + lg2<(N >> 1)>();
}

template <>
__host__ __device__ inline uint32_t lg2<0>() {
  return 0;
}

template <>
__host__ __device__ inline uint32_t lg2<1>() {
  return 0;
}

struct Ray4 {
  float4 origin;
  float4 dir;
};

struct Aabb4 {
  float4 min;
  float4 max;
};

std::ostream& operator<<(std::ostream& os, const float4& x) {
  os << x.x << "  " << x.y << " " << x.z << " " << x.w;
  return os;
}

std::ostream& operator<<(std::ostream& os, const int4& x) {
  os << x.x << "  " << x.y << " " << x.z << " " << x.w;
  return os;
}

std::ostream& operator<<(std::ostream& os, const float3& x) {
  os << x.x << "  " << x.y << " " << x.z << " ";
  return os;
}

std::ostream& operator<<(std::ostream& os, const int3& x) {
  os << x.x << "  " << x.y << " " << x.z << " ";
  return os;
}

std::ostream& operator<<(std::ostream& os, const Ray& r) {
  float3 origin = make_float3(r.ox, r.oy, r.oz);
  float3 dir = make_float3(r.dx, r.dy, r.dz);
  os << "o = " << origin << " tmin = " << r.tmin << " d = " << dir
     << " tmax = " << r.tmax;
  return os;
}

inline __host__ __device__ float4 cross(const float4& a, const float4& b) {
  return make_float4(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
                     a.x * b.y - a.y * b.x, 0.0f);
}

inline __host__ __device__ float dot43(const float4& a, const float4& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

template <typename SourceType, typename DestinationType>
void __host__ __device__ assign(const SourceType& source,
                                DestinationType* dest) {}

template <>
void __host__ __device__ assign<float3, float4>(const float3& source,
                                                float4* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
  dest->w = 0.0f;
}

template <>
void __host__ __device__ assign<int3, int4>(const int3& source, int4* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
  dest->w = 0.0f;
}

template <>
void __host__ __device__ assign<float4, float3>(const float4& source,
                                                float3* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
}

template <>
void __host__ __device__ assign<int4, int3>(const int4& source, int3* dest) {
  dest->x = source.x;
  dest->y = source.y;
  dest->z = source.z;
}

struct timespec getRealTime() {
  struct timespec ts;
#ifdef __FreeBSD__
  clock_gettime(CLOCK_MONOTONIC, &ts);  // Works on FreeBSD
#else
  clock_gettime(CLOCK_REALTIME, &ts);
#endif
  return ts;
}

template <typename T>
inline __device__ __host__ const T* RunTimeSelect(bool condition,
                                                  const T* trueResult,
                                                  const T* falseResult) {
  const uintptr_t c = condition * ~(static_cast<uintptr_t>(0x0));
  return reinterpret_cast<const T*>(
      ((reinterpret_cast<uintptr_t>(trueResult) & c) |
       (reinterpret_cast<uintptr_t>(falseResult) & ~c)));
}

template <typename T>
inline __device__ __host__ void RunTimeAssignIf(bool condition, T* dest,
                                                const T* src) {
  T dummy;
  const uintptr_t c = condition * ~(static_cast<uintptr_t>(0x0));
  *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(dest) & c) |
                         (reinterpret_cast<uintptr_t>(&dummy) & ~c))) = *src;
}

double getTimeDiffMs(const struct timespec& start, const struct timespec& end) {
  // start: X s, A ns
  // end:   Y s, B ns
  // (Y - (X + 1)) * 1000000.0 + B / 1000.0 + 1000000.0 - A / 1000.0
  // = (Y - X) * 1000000.0 - 1000000.0 + B / 1000.0 + 1000000.0 - A / 1000.0
  // = (Y - X) * 1000000.0 + B / 1000.0 - A / 1000.0
  double microsecond_diff = 1000000.0 * (end.tv_sec - start.tv_sec) +
                            end.tv_nsec / 1000.0 - start.tv_nsec / 1000.0;
  return microsecond_diff;
}

__global__ void createRaysOrthoKernel(int width, int height, float x0, float y0,
                                      float z, float dx, float dy,
                                      float4* d_rays) {
  int rayx = threadIdx.x + blockIdx.x * blockDim.x;
  int rayy = threadIdx.y + blockIdx.y * blockDim.y;
  if (rayx >= width || rayy >= height) return;

  int idx = rayx + rayy * width;
  d_rays[2 * idx + 0] =
      make_float4(x0 + rayx * dx, y0 + rayy * dy, z, 0);  // origin, tmin
  d_rays[2 * idx + 1] = make_float4(0, 0, 1, 1e34f);      // dir, tmax
}

#ifdef UPDATE_HITS_SOA
__global__ __launch_bounds__(THREADS_PER_BLOCK,
                             MIN_BLOCKS) void reorderHitsKernel(Hit* hits,
                                                                int numRays) {
  __shared__ Hit localHits[THREADS_PER_BLOCK];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < numRays) {
    float* t_values = reinterpret_cast<float*>(hits + blockIdx.x * blockDim.x);
    int* triIds = reinterpret_cast<int*>(t_values + blockDim.x);
    float* u_values = reinterpret_cast<float*>(triIds + blockDim.x);
    float* v_values = u_values + blockDim.x;
    localHits[threadIdx.x].t = t_values[threadIdx.x];
    localHits[threadIdx.x].triId = triIds[threadIdx.x];
    localHits[threadIdx.x].u = u_values[threadIdx.x];
    localHits[threadIdx.x].v = v_values[threadIdx.x];
  }
  __syncthreads();
  if (tid < numRays) {
    hits[threadIdx.x + blockIdx.x * blockDim.x] = localHits[threadIdx.x];
  }
}
#endif

#define DIVERGENCE_FREE_CHILD_BOUNDS
inline __device__ __host__ Aabb4 getChildBounds(const Aabb4& bounds,
                                                const float4& center,
                                                unsigned char octant) {
  Aabb4 result;
  float4 min = bounds.min;
  float4 max = bounds.max;
#ifdef DIVERGENCE_FREE_CHILD_BOUNDS
  const float4* min_center[2] = {&min, &center};
  const float4* center_max[2] = {&center, &max};
#endif

#ifdef DIVERGENCE_FREE_CHILD_BOUNDS
  unsigned char xBit = (octant >> 0) & 0x1;
  unsigned char yBit = (octant >> 1) & 0x1;
  unsigned char zBit = (octant >> 2) & 0x1;
  min.x = min_center[xBit]->x;
  max.x = center_max[xBit]->x;
  min.y = min_center[yBit]->y;
  max.y = center_max[yBit]->y;
  min.z = min_center[zBit]->z;
  max.z = center_max[zBit]->z;
#else
  min.x = ((octant & (0x1 << 0)) > 0 ? center.x : min.x);
  max.x = ((octant & (0x1 << 0)) > 0 ? max.x : center.x);
  min.y = ((octant & (0x1 << 1)) > 0 ? center.y : min.y);
  max.y = ((octant & (0x1 << 1)) > 0 ? max.y : center.y);
  min.z = ((octant & (0x1 << 2)) > 0 ? center.z : min.z);
  max.z = ((octant & (0x1 << 2)) > 0 ? max.z : center.z);
#endif
  result.min = min;
  result.max = max;
  return result;
}

inline __device__ __host__ bool isValidT(float t, float t_near, float t_far) {
  return !isnan(t) & t < t_far & t >= t_near;
}

template <typename T>
inline __device__ __host__ void exchangeIf(bool condition, T* temp, T* x,
                                           T* y) {
  uintptr_t c = condition;
  c -= 1;
  *temp = *x;
  *x = *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(x) & c) |
                              (reinterpret_cast<uintptr_t>(y) & ~c)));
  *y = *reinterpret_cast<T*>(((reinterpret_cast<uintptr_t>(temp) & ~c) |
                              (reinterpret_cast<uintptr_t>(y) & c)));
}

template <>
inline __device__ __host__ void exchangeIf<unsigned char>(bool condition,
                                                          unsigned char* temp,
                                                          unsigned char* x,
                                                          unsigned char* y) {
  unsigned char c = condition;
  c -= 1;
  *temp = *x;
  *x = ((((*x) & c) | ((*y) & ~c)));
  *y = (((*temp) & ~c) | ((*y) & c));
}

//#define USE_COALESCED_HIT_UPDATE
inline __device__ __host__ void updateHitBuffer(Hit* closest, Hit* hitBuf) {
#ifdef USE_COALESCED_HIT_UPDATE
  unsigned char* out = reinterpret_cast<unsigned char*>(hitBuf);
  uchar4 c0 = *reinterpret_cast<const uchar4*>(&closest.t);
  uchar4 c4 = *reinterpret_cast<const uchar4*>(&closest.triId);
  uchar4 c8 = *reinterpret_cast<const uchar4*>(&closest.u);
  uchar4 c12 = *reinterpret_cast<const uchar4*>(&closest.v);
  out[0] = c0.x;
  out[1] = c4.y;
  out[2] = c8.z;
  out[3] = c12.w;
#else
  hitBuf->t = closest->t;
  hitBuf->triId = closest->triId;
  hitBuf->u = closest->u;
  hitBuf->v = closest->v;
#endif
}

__device__ __inline__ float min4(float a, float b, float c, float d) {
  return fminf(fminf(fminf(a, b), c), d);
}

__device__ __inline__ float max4(float a, float b, float c, float d) {
  return fmaxf(fmaxf(fmaxf(a, b), c), d);
}

inline __device__ bool intersectAabb2(const float4& origin,
                                      const float4& invDirection,
                                      const Aabb4& bounds, float t0, float t1,
                                      float* tNear, float* tFar) {
  const float4 ood =
      make_float4(origin.x * invDirection.x, origin.y * invDirection.y,
                  origin.z * invDirection.z, 0.0f);
  const float4& min_bounds = bounds.min;
  const float4& max_bounds = bounds.max;
  float4 min_bounds_diff =
      make_float4(min_bounds.x - origin.x, min_bounds.y - origin.y,
                  min_bounds.z - origin.z, 0.0f);
  float4 tmins = make_float4(min_bounds_diff.x * invDirection.x,
                             min_bounds_diff.y * invDirection.y,
                             min_bounds_diff.z * invDirection.z, 0.0f);
  float4 max_bounds_diff =
      make_float4(max_bounds.x - origin.x, max_bounds.y - origin.y,
                  max_bounds.z - origin.z, 0.0f);
  float4 tmaxs = make_float4(max_bounds_diff.x * invDirection.x,
                             max_bounds_diff.y * invDirection.y,
                             max_bounds_diff.z * invDirection.z, 0.0f);
  float tminbox = max4(t0, fminf(tmins.x, tmaxs.x), fminf(tmins.y, tmaxs.y),
                       fminf(tmins.z, tmaxs.z));
  float tmaxbox = min4(t1, fmaxf(tmins.x, tmaxs.x), fmaxf(tmins.y, tmaxs.y),
                       fmaxf(tmins.z, tmaxs.z));
  bool intersect = (tminbox <= tmaxbox);
  *tNear = tminbox;
  *tFar = tmaxbox;
  return intersect;
}

#define DIVERGENCE_FREE_INSTERSECT_TRIANGLE
inline __device__ bool intersectTriangle(const float4& origin,
                                         const float4& dir, const int4* indices,
                                         const float4* vertices, int triId,
                                         Hit& isect, int numTriangles,
                                         int numVertices) {
  const int4 tri = indices[triId];
  /*const int4 tri = tex1Dfetch(texture_indices, triId);*/
  /*const float4 a = vertices[tri.x];*/
  /*const float4 b = vertices[tri.y];*/
  /*const float4 c = vertices[tri.z];*/
  const float4 a = tex1Dfetch(texture_vertices, tri.x);
  const float4 b = tex1Dfetch(texture_vertices, tri.y);
  const float4 c = tex1Dfetch(texture_vertices, tri.z);
  const float4 e1 = b - a;
  const float4 e2 = c - a;
  const float4 pVec =
      make_float4(dir.y * e2.z - dir.z * e2.y, dir.z * e2.x - dir.x * e2.z,
                  dir.x * e2.y - dir.y * e2.x, 0.0f);
  float det = dot43(e1, pVec);
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (det > -kEpsilon && det < kEpsilon) return false;
#endif
  float invDet = 1.0f / det;
  float4 tVec =
      make_float4(origin.x - a.x, origin.y - a.y, origin.z - a.z, 0.0f);
  float4 qVec =
      make_float4(tVec.y * e1.z - tVec.z * e1.y, tVec.z * e1.x - tVec.x * e1.z,
                  tVec.x * e1.y - tVec.y * e1.x, 0.0f);
  float t = e2.x * qVec.x;
  t += e2.y * qVec.y;
  t += e2.z * qVec.z;
  t *= invDet;
// Do not allow ray origin in front of triangle
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (t < 0.0f) return false;
#endif
  float u = tVec.x * pVec.x;
  u += tVec.y * pVec.y;
  u += tVec.z * pVec.z;
  u *= invDet;
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (u < 0.0f || u > 1.0f) return false;
#endif
  float v = dir.x * qVec.x;
  v += dir.y * qVec.y;
  v += dir.z * qVec.z;
  v *= invDet;
#ifndef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  if (v < 0.0f || u + v > 1.0f) return false;
#endif
  isect.t = t;
  isect.triId = triId;
  isect.u = u;
  isect.v = v;
#ifdef DIVERGENCE_FREE_INSTERSECT_TRIANGLE
  return t >= 0.0f & u >= 0.0f & u <= 1.0f & v >= 0.0f & ((u + v) <= 1.0f);
#else
  return true;
#endif
}

template <typename I, typename T>
inline __device__ __host__ void permute3(const I* order, T* a) {
  T tempT;
  I tempOrder[3] = {order[0], order[1], order[2]};
  I tempI;
  bool lessThan = tempOrder[2] < tempOrder[1];
  exchangeIf(lessThan, &tempT, a + 1, a + 2);
  exchangeIf(lessThan, &tempI, tempOrder + 1, tempOrder + 2);
  lessThan = tempOrder[1] < tempOrder[0];
  exchangeIf(lessThan, &tempT, a, a + 1);
  exchangeIf(lessThan, &tempI, tempOrder, tempOrder + 1);
  lessThan = tempOrder[2] < tempOrder[1];
  exchangeIf(lessThan, &tempT, a + 1, a + 2);
  exchangeIf(lessThan, &tempI, tempOrder + 1, tempOrder + 2);
};

template <typename T, typename Comparator>
inline __device__ __host__ void compareExchange(const Comparator& c, T* temp,
                                                T* x, T* y) {
  bool lessThan = c(*y, *x);
  exchangeIf(lessThan, temp, x, y);
}

template <typename T, typename Comparator>
inline __device__ __host__ void sort3(const Comparator& c, T* a) {
  T temp;
  compareExchange(c, &temp, a + 1, a + 2);
  compareExchange(c, &temp, a, a + 1);
  compareExchange(c, &temp, a + 1, a + 2);
};

inline __host__ __device__ __host__ void createEvents(
    const float3* origin, const float3* direction, const float4& invDirection,
    const float4& center, const float4& hit, float tNear, float tFar,
    OctreeEvent* events, int16_t* N) {
  // Compute the default entry point.
  unsigned char xBit = (hit.x > center.x);
  unsigned char yBit = (hit.y > center.y);
  unsigned char zBit = (hit.z > center.z);
  unsigned char octant = xBit | (yBit << 1) | (zBit << 2);

  // Compute the t values for which the ray crosses each x, y, z intercept.
  float4 diff_center_origin = make_float4(
      center.x - origin->x, center.y - origin->y, center.z - origin->z, 0.0f);
  float4 t = make_float4(diff_center_origin.x * invDirection.x,
                         diff_center_origin.y * invDirection.y,
                         diff_center_origin.z * invDirection.z, 0.0f);

  // Create the events, unsorted.
  OctreeEvent eventEntry = {OCTREE_EVENT_ENTRY, octant, tNear};
  OctreeEvent eventX = {OCTREE_EVENT_X, 0x1, t.x};
  OctreeEvent eventY = {OCTREE_EVENT_Y, 0x2, t.y};
  OctreeEvent eventZ = {OCTREE_EVENT_Z, 0x4, t.z};
  OctreeEvent eventExit = {OCTREE_EVENT_EXIT, 0, tFar};

  events[1] = eventX;
  events[2] = eventY;
  events[3] = eventZ;

  OctreeEvent* planarEvents = &events[1];

  // Mask lookup table.

  // Each event is compared and a value for each mask output is computed.
  // This is necessary since some events could have equal t-intercepts.
  // This means the ray hits some projection of the centroid in the x, y, z
  // planes defined by the centroid: it hits (x, y, z), (x, y), (y, x),
  // or (y, z).
  //
  // NOTE: if there is a case such as t_x == t_y and t_y == t_z, but
  // t_x != t_z, this is treated as if t_x = t_y = t_z.
  unsigned char equals_01 = (planarEvents[0].t == planarEvents[1].t);
  unsigned char equals_02 = (planarEvents[0].t == planarEvents[2].t);
  unsigned char equals_12 = (planarEvents[1].t == planarEvents[2].t);
  unsigned char unique0 = (equals_01 | equals_02) ^ 0x1;
  unsigned char unique1 = (equals_01 | equals_12) ^ 0x1;
  unsigned char unique2 = (equals_02 | equals_12) ^ 0x1;

  // Result of unique_* is:
  // unique0 unique1 unique2   output index
  //    0       0       0          000 = 0
  //    0       0       1          001 = 1
  //    0       1       0          010 = 2
  //    1       0       0          011 = 3
  //    1       1       1          100 = 4
  //
  // The mask lookup index in binary is: u'_2 u'_1 u'_0.
  // This results in the sorted order of the output unique values and give
  // an index that can be used to acces the mask lookup table.
  // The equations for the u'_* values are:
  //
  // u'_2 = u_2 && u_1
  // u'_1 = (u_2 || u_1) && !u_0
  // u'_0 = (u_2 || u_0) && !u_1
  unsigned char maskLookupIndex = ((unique0 & unique1) << 2) |
                                  ((unique0 | unique1) & (unique2 ^ 0x1)) << 1 |
                                  ((unique2 | unique0) & (unique1 ^ 0x1));
  const unsigned char mask_X = 0x1;
  const unsigned char mask_Y = 0x2;
  const unsigned char mask_Z = 0x4;
  const unsigned char mask_XY = mask_X | mask_Y;
  const unsigned char mask_XZ = mask_X | mask_Z;
  const unsigned char mask_YZ = mask_Y | mask_Z;
  const unsigned char mask_XYZ = mask_X | mask_Y | mask_Z;
  const unsigned char maskLookupTable[5][3] = {
      {mask_XYZ, mask_XYZ, mask_XYZ},  // 000 - t_x, t_y, t_z all equal
      {mask_XY, mask_XY, mask_Z},      // 001 - t_x == t_y only
      {mask_XZ, mask_Y, mask_XZ},      // 010 - t_x == t_z only
      {mask_X, mask_YZ, mask_YZ},      // 100 - t_y == t_z only
      {mask_X, mask_Y, mask_Z}         // 111 - t_x, t_y, t_z all unique
  };

  // Permutation lookup table and permutation index.
  // After sorting, validity is checked, and invalid entries must be permuted.
  // The permutation is gotten by computing an index into a lookup table.
  unsigned char check0 = isValidT(planarEvents[0].t, tNear, tFar);
  unsigned char check1 = isValidT(planarEvents[1].t, tNear, tFar);
  unsigned char check2 = isValidT(planarEvents[2].t, tNear, tFar);
  // After validity check, need to check both uniqueness and validity to ensure
  // each element is valid.  This is a conservative evaluation: if A and B are
  // equal and one is invalid, then both are invalid.
  unsigned char check01 = unique0 | unique1 | (check0 & check1);
  unsigned char check02 = unique0 | unique2 | (check0 & check2);
  unsigned char check12 = unique1 | unique2 | (check1 & check2);
  // Final validity computed.  A is valid if the initial check is true
  // and it is not equivalent to some other invalid value.
  unsigned char validTable[3];
  validTable[0] = check0 & check01 & check02;
  validTable[1] = check1 & check01 & check12;
  validTable[2] = check2 & check02 & check12;
  const unsigned char permutationTable[8][3] = {
      {0, 1, 2},  // 000
      {1, 2, 0},  // 001
      {1, 0, 2},  // 010
      {2, 0, 1},  // 011
      {0, 1, 2},  // 100
      {0, 2, 1},  // 101
      {0, 1, 2},  // 110
      {0, 1, 2}   // 111
  };

  // Compute masks according to table.
  planarEvents[0].mask = maskLookupTable[maskLookupIndex][0];
  planarEvents[1].mask = maskLookupTable[maskLookupIndex][1];
  planarEvents[2].mask = maskLookupTable[maskLookupIndex][2];

  // Sort.
  OctreeEventComparator comparator;
  sort3(comparator, planarEvents);

  // Compute the permutation index here.
  // The index computation needs to be delayed, since the permutation table
  // assumes events are in sorted order w.r.t t-values.
  unsigned char permutationIndex = validTable[planarEvents[2].type] |
                                   validTable[planarEvents[1].type] << 1 |
                                   validTable[planarEvents[0].type] << 2;
  // Shuffle according to table. Events that are duplicate or invalid
  // will be shuffled to the end.  This is why sorted order is important.
  permute3(permutationTable[permutationIndex], planarEvents);

  // Compute number of internal events.
  // Count 0 if valid_0
  // Count 1 if:
  //    valid_1 unique0 unique1 unique2 output
  //      1         0       0       0       0
  //                1       0       0       1
  //                0       1       0       1
  //                0       0       1       0
  //                1       1       1       1
  unsigned char k = validTable[0] + (validTable[1] & (unique0 | unique1)) +
                    (validTable[2] & unique2);

  // Write entry and exit events.
  events[0] = eventEntry;
  events[k + 1] = eventExit;

  // Number of events total (including entry and exit).
  int16_t Nevents = k + 2;
  *N = Nevents;

  // Compute entry mask. This should usually be 000, but if the ray hits
  // an X, Y, or Z plane at the boundary of a node, then the mask needs
  // to be different.  Computing the XOR of this mask with the bitwise
  // representation of the octant gives the correct entry mask.
  bool entryEqualsFirst = events[0].t == events[1].t;
  bool isX = events[1].type == OCTREE_EVENT_X;
  bool isY = events[1].type == OCTREE_EVENT_Y;
  bool isZ = events[1].type == OCTREE_EVENT_Z;
  unsigned char xMask =
      Nevents > 2 & entryEqualsFirst & isX & (xBit | (direction->x < 0.0f));
  unsigned char yMask =
      Nevents > 2 & entryEqualsFirst & isY & (yBit | (direction->y < 0.0f));
  unsigned char zMask =
      Nevents > 2 & entryEqualsFirst & isZ & (zBit | (direction->z < 0.0f));
  unsigned char mask = xMask | (yMask << 1) | (zMask << 2);
  events[0].mask = events[0].mask ^ mask;
}

inline __host__ __device__ __host__ void createEvents0(
    const float4& origin, const float4& direction, const float4& invDirection,
    const float4& center, const float4& hit, float tNear, float tFar,
    OctreeEvent* events, int16_t* N) {
  float4 diff_center_origin = make_float4(
      center.x - origin.x, center.y - origin.y, center.z - origin.z, 0.0f);
  float4 t = make_float4(diff_center_origin.x * invDirection.x,
                         diff_center_origin.y * invDirection.y,
                         diff_center_origin.z * invDirection.z, 0.0f);
  // Create the events, unsorted.
  events[1].type = OCTREE_EVENT_X;
  events[1].mask = 0x1;
  events[1].t = t.x;
  events[2].type = OCTREE_EVENT_Y;
  events[2].mask = 0x2;
  events[2].t = t.y;
  events[2].type = OCTREE_EVENT_Z;
  events[2].mask = 0x4;
  events[2].t = t.z;
  // Sort the planarEvents, so we can implement a front-to-back traversal.
  exchangeIf(
      !isValidT(events[2].t, tNear, tFar) |
          (events[2].t > events[3].t & isValidT(events[3].t, tNear, tFar)),
      &events[0], &events[2], &events[3]);
  exchangeIf(
      !isValidT(events[1].t, tNear, tFar) |
          (events[1].t > events[2].t & isValidT(events[2].t, tNear, tFar)),
      &events[0], &events[1], &events[2]);
  exchangeIf(
      !isValidT(events[2].t, tNear, tFar) |
          (events[2].t > events[3].t & isValidT(events[3].t, tNear, tFar)),
      &events[0], &events[2], &events[3]);
  // Discard planarEvents with t > tFar.
  // k is the index of the last event.
  int k = 2;
  while (k >= 0 && !isValidT(events[k + 1].t, tNear, tFar)) --k;
  // Consolidate planarEvents that have the same t-value.
  // There are only 1, 2, or 3 planarEvents, so we just explicitly compute
  // this.
  if (k == 2) {
    bool left_equal = (events[1].t == events[2].t);
    bool right_equal = (events[2].t == events[3].t);
    if (left_equal && right_equal) {
      events[1].mask = events[1].mask | events[2].mask | events[3].mask;
      k = 0;
    } else if (left_equal) {
      events[1].mask = events[1].mask | events[2].mask;
      events[2] = events[3];
      k = 1;
    } else if (right_equal) {
      events[2].mask = events[2].mask | events[3].mask;
      k = 1;
    }
  } else if (k == 1) {
    if (events[1].t == events[2].t) {
      events[1].mask = events[1].mask | events[2].mask;
      k = 0;
    }
  }
  unsigned char xBit = (hit.x > center.x);
  unsigned char yBit = (hit.y > center.y);
  unsigned char zBit = (hit.z > center.z);
  events[0].type = OCTREE_EVENT_ENTRY;
  events[0].t = tNear;
  events[0].mask = xBit | (yBit << 1) | (zBit << 2);
  events[k + 2].type = OCTREE_EVENT_EXIT;
  events[k + 2].t = tFar;
  events[k + 2].mask = 0;
  *N = (k + 1) + 2;
  unsigned char xMask =
      (events[1].type == OCTREE_EVENT_X) & ((xBit == 1) | (direction.x < 0.0f));
  unsigned char yMask =
      (events[1].type == OCTREE_EVENT_Y) & ((yBit == 1) | (direction.y < 0.0f));
  unsigned char zMask =
      (events[1].type == OCTREE_EVENT_Z) & ((zBit == 1) | (direction.z < 0.0f));
  unsigned char mask = xMask | (yMask << 1) | (zMask << 2);
  //  if ((k + 1) + 2 > 2 && events[0].t == events[1].t)
  events[0].mask =
      events[0].mask ^ (((k + 1) + 2 > 2 && events[0].t == events[1].t) * mask);
}

inline float4 __device__ __host__ getSplitPoint(const OctNode128* node,
                                                const Aabb4& bounds) {
  const float4& min = bounds.min;
  const float4& max = bounds.max;
  uint16_t num_samples = node->footer.internal.sizeDescriptor;
  if (num_samples <= 1) return 0.5f * (min + max);
  float inv = 1.0 / (num_samples - 1);
  float4 step_size = inv * (max - min);
  float4 split_point =
      make_float4(node->footer.internal.i, node->footer.internal.j,
                  node->footer.internal.k, 0.0f);
  split_point *= step_size;
  split_point += min;
  return split_point;
}

//#define DEBUG_TRAVERSE
#ifdef DEBUG_TRAVERSE
//#define DEBUG_TRAVERSE_THREAD_ID 389308
#define DEBUG_TRAVERSE_THREAD_ID 190365
//#define DEBUG_TRAVERSE_THREAD_ID 386858  // t = 0.485482
#endif
#define MAX_DEPTH 15
#define MAX_EVENTS 4
#define STACK_SIZE (MAX_EVENTS * MAX_DEPTH)
#ifdef USE_PERSISTENT

#endif
inline __device__ void intersectOctree(
    const float4* rays, const uint4* nodes, const float4* vertices,
    const int4* indices, const uint32_t* references, const Aabb4 bounds,
    uint32_t numTriangles, uint32_t numVertices, uint32_t numRays, Hit* hits) {
  // NOTE:
  //    1) We need to examine 4 nodes per octree node.
  //    4) Because of (1), we create a stack of size:
  //          4 * d  * B
  //    where we need B bytes per node to store a reference on the stack.
  // Here B = 4, since unsigned ints will be used.
  //
  // NOTE: With treelet demarcations, we could allow treelets
  // of maximum size 16k nodes, so short ints could be use where B_16 = 2.
  //
  // NOTE: This uses thread-local storage - it is really global memory as
  // opposed to shared memory.  The danger of using shared memory is that
  // many threads may fetch the same node, so it might be best to let
  // the GPU manage the cache on its own and hopefully we only fetch
  // each node that we actually need.
  const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  const uint32_t warpId = tid / WARP_SIZE;       // get our warpId
  const unsigned char laneId = tid % WARP_SIZE;  // get our warp index
  const uint32_t warpIdx = warpId % WARPS_PER_BLOCK;
  int nodeIdStack[STACK_SIZE];
  Aabb4 aabbStack[STACK_SIZE];
  float tNearStack[STACK_SIZE];
  float tFarStack[STACK_SIZE];
  __shared__ volatile int localRayCount[WARPS_PER_BLOCK];
  __shared__ volatile int localNextRay[WARPS_PER_BLOCK];

  localNextRay[warpIdx] = 0;
  localRayCount[warpIdx] = 0;

  do {
    // If we are the first thread in the warp, check our work status
    // and add more work if needed.
    if (laneId == 0 && localRayCount[warpIdx] <= 0) {
      localNextRay[warpIdx] = atomicAdd(&nextRayIndex, WARP_BATCH_SIZE);
      localRayCount[warpIdx] = WARP_BATCH_SIZE;
    }

    // Get the next ray for this thread.
    int rayIdx = localNextRay[warpIdx] + laneId;
#ifdef DEBUG_TRAVERSE
    int numNodes = 0;
    int numLeaves = 0;
    int depthStack[STACK_SIZE];
    int depth = 0;
    depthStack[0] = 0;
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      const float4 o = rays[rayIdx * 2];
      const float4 d = rays[rayIdx * 2 + 1];
      printf("[%d] o = %f %f %f, d = %f %f %f\n", tid, o.x, o.y, o.z, d.x, d.y,
             d.z);
    }
#endif
    bool goodThread = rayIdx < numRays;
    if (!goodThread) break;

    // Update counts and next ray to get.
    if (laneId == 0) {
      localNextRay[warpIdx] += WARP_SIZE;
      localRayCount[warpIdx] -= WARP_SIZE;
    }

    /**reinterpret_cast<float4*>(&localRays[threadIdx.x]) =*/
    /*tex1Dfetch(texture_rays, rayIdx * 2);*/
    /**(reinterpret_cast<float4*>(&localRays[threadIdx.x]) + 1) =*/
    /*tex1Dfetch(texture_rays, rayIdx * 2 + 1);*/

    // Initialize traversal.
    const float4 origin = rays[rayIdx * 2];
    const float4 dir = rays[rayIdx * 2 + 1];
    const float4 invDirection =
        make_float4(1.0f / dir.x, 1.0f / dir.y, 1.0f / dir.z, 0.0f);
    int16_t stackEnd = 1;
    int currentId = -1;
    float tNear = 0.0f, tFar = 0.0f;
    bool stackEmpty = false;
    bool objectHit = false;

    Hit closest;
    closest.t = NPP_MAXABS_32F;
    closest.triId = -1;

    // Put the root onto the stack.
    nodeIdStack[0] = 0;
    aabbStack[0] = bounds;
    bool hitBounds =
        intersectAabb2(origin, invDirection, aabbStack[0], 0.0f, NPP_MAXABS_32F,
                       &tNearStack[0], &tFarStack[0]);
#ifdef DEBUG_TRAVERSE
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      printf(
          "hitBounds = %d, objectHit = %d, stackEmpty = %d tNear = % f tFar = "
          "% f\n",
          hitBounds, objectHit, stackEmpty, tNearStack[0], tFarStack[0]);
    }
#endif

    while (hitBounds & !(objectHit | stackEmpty)) {
      // Setup beore entering loop.
      stackEmpty = (stackEnd <= 0);
      currentId = nodeIdStack[!stackEmpty * (stackEnd - 1)];
      OctNode128 currentNode;
      if (!stackEmpty) {
        *reinterpret_cast<uint4*>(&currentNode) =
            tex1Dfetch(texture_nodes, currentId);
        /*currentNode = *reinterpret_cast<const
         * OctNode128*>(&nodes[currentId]);*/
      }
      bool foundLeaf = (currentNode.header.type == NODE_LEAF) && !stackEmpty;
      tNear = !stackEmpty * tNearStack[!stackEmpty * (stackEnd - 1)];
      tFar = !stackEmpty * tFarStack[!stackEmpty * (stackEnd - 1)];

      // Go until stack empty or found a leaf.
      while (!foundLeaf && !stackEmpty) {
        // Get node information.
        currentId = nodeIdStack[stackEnd - 1];
        Aabb4 currentBounds = aabbStack[stackEnd - 1];
#ifdef DEBUG_TRAVERSE
        ++numNodes;
        depth = depthStack[stackEnd - 1];
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          printf("[N $%x #%d @%d +%d %d, %d, %d, %d] %f %f\n",
                 currentNode.footer.internal.childMask,
                 countBits(currentNode.footer.internal.childMask),
                 currentNode.header.octant, currentNode.header.offset,
                 currentNode.footer.internal.i, currentNode.footer.internal.j,
                 currentNode.footer.internal.k,
                 currentNode.footer.internal.sizeDescriptor, tNear, tFar);
        }
#endif
        float4 hit =
            make_float4(origin.x + tNear * dir.x, origin.y + tNear * dir.y,
                        origin.z + tNear * dir.z, 0.0f);
        float4 center = getSplitPoint(&currentNode, currentBounds);

        //  Get the events, in order of they are hit.
        int16_t numEvents = 0;
        OctreeEvent events[5];
        int16_t numValidEvents = 0;
        createEvents0(origin, dir, invDirection, center, hit, tNear, tFar,
                      events, &numEvents);
        unsigned char octantBits = 0x0;

        // Get children.
        uint32_t children[8];
        uint32_t childId = currentNode.header.offset;
        octantBits = currentNode.footer.internal.childMask;
#pragma unroll
        for (uint32_t i = 0; i < 8; ++i) {
          children[i] = childId;
          childId += ((octantBits >> i) & 0x1);
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {*/
/*printf("%d ", children[i]);*/
/*}*/
#endif
        }
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {*/
/*printf("\n");*/
/*}*/
#endif
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          int numChildren = countBits(octantBits);
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          for (int i = 0; i < numChildren; ++i) {
            OctNode128 child = *reinterpret_cast<const OctNode128*>(
                &nodes[currentNode.header.offset + i]);
            if (child.header.type == NODE_LEAF) {
              printf("L %d %d %d, ", child.header.octant, child.header.offset,
                     child.footer.leaf.size);
            } else {
              printf("N %d, ", child.header.octant);
            }
          }
          printf("\n");
        }
#endif

        // Figure which octants were hit are non-empty.
        unsigned char octant = 0x0;
        numValidEvents = 0;
        for (int16_t i = 0; i < numEvents - 1; ++i) {
          octant = octant ^ events[i].mask;
          bool hasChild = ((octantBits & (0x1 << octant)) != 0);
          numValidEvents += hasChild;
        }
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          octant = 0x0;
          printf("[%08d] #%d", currentId, numEvents);
          for (int i = 0; i < depth; ++i) printf("  ");
          for (int i = 0; i < numEvents - 1; ++i) {
            octant = octant ^ events[i].mask;
            printf("(%d, %f, %x, %d) ", octant, events[i].t, events[i].mask,
                   ((octantBits & (0x1 << octant)) != 0));
          }
          printf("\n");
        }
#endif

#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("add -->");
        }
#endif
        // Add the children in reverse order of being hit to the stack.  This
        // way,  the child that was hit first gets popped first.
        int16_t k = -1;  // keep track of which valid event we have
        octant = 0x0;
        for (int16_t i = 0; (i < numEvents - 1) & ((k + 1) < numValidEvents);
             ++i) {
          octant = octant ^ events[i].mask;
          bool hasChild = ((octantBits & (0x1 << octant)) != 0);
          k += hasChild;
          int16_t nextStack = (stackEnd - 1) + numValidEvents - k - 1;
          if (hasChild) {  // divergence
#ifdef DEBUG_TRAVERSE
            if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
              printf("%d ", children[octant]);
            }
#endif
            nodeIdStack[nextStack] = children[octant];
            aabbStack[nextStack] =
                getChildBounds(currentBounds, center, octant);
            tNearStack[nextStack] = events[i].t;
            tFarStack[nextStack] = events[i + 1].t;
#ifdef DEBUG_TRAVERSE
            depthStack[nextStack] = depth + 1;
#endif
          }
        }
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("\n");
        }
#endif
        stackEnd += numValidEvents;
        --stackEnd;
        stackEmpty = (stackEnd <= 0);
        currentId = nodeIdStack[!stackEmpty * (stackEnd - 1)];
        if (!stackEmpty) {
          *reinterpret_cast<uint4*>(&currentNode) =
              tex1Dfetch(texture_nodes, currentId);
          /*currentNode = *reinterpret_cast<const
           * OctNode128*>(&nodes[currentId]);*/
        }
        foundLeaf = (currentNode.header.type == NODE_LEAF) && !stackEmpty;
        tNear = !stackEmpty * tNearStack[!stackEmpty * (stackEnd - 1)];
        tFar = !stackEmpty * tFarStack[!stackEmpty * (stackEnd - 1)];
      }  // end of while (!foundLeaf && !stackEmpty)
#ifdef DEBUG_TRAVERSE
      if (foundLeaf) {
        ++numLeaves;
        depth = depthStack[stackEnd - 1];
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
          printf("[%08d]", currentId);
          for (int i = 0; i < depth; ++i) printf("  ");
          uint32_t size = currentNode.footer.leaf.size;
          uint32_t offset = currentNode.header.offset;
          uint32_t octant = currentNode.header.octant;
          printf("[L #%d @%d +%d] %f %f\n", size, octant, offset, tNear, tFar);
        }
      }
#endif

      // We either have a leaf or stack is empty.
      uint32_t numPrimitives = currentNode.footer.leaf.size;
      uint32_t offset = currentNode.header.offset;
      bool triangleHit = false;
#ifdef DEBUG_TRAVERSE
/*if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {*/
/*printf("-->[L #%d @%d +%d]\n", numPrimitives, octant, offset);*/
/*}*/
#endif

#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("[%08d]", currentId);
        for (int i = 0; i < depth; ++i) printf("  ");
      }
#endif
      numPrimitives *= !stackEmpty;
      offset *= !stackEmpty;
      for (uint32_t i = 0; i < numPrimitives; ++i) {
        uint32_t triId = references[i + offset];
        /*uint32_t triId = tex1Dfetch(texture_references, i + offset);*/
        Hit isect;
        isect.t = NPP_MAXABS_32F;
        isect.triId = -1;
        bool isNewClosest =
            intersectTriangle(origin, dir, indices, vertices, triId, isect,
                              numTriangles, numVertices) &&
            isect.t >= tNear && isect.t <= tFar && isect.t < closest.t;
        if (isNewClosest) closest = isect;
        /*closest.t = isNewClosest * isect.t + !isNewClosest * closest.t;*/
        /*closest.triId =*/
        /*isNewClosest * isect.triId + !isNewClosest * closest.triId;*/
        /*closest.u = isNewClosest * isect.u + !isNewClosest * closest.u;*/
        /*closest.v = isNewClosest * isect.v + !isNewClosest * closest.v;*/
        triangleHit = isNewClosest || triangleHit;
#ifdef DEBUG_TRAVERSE
        if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
          printf("(%f, %d)", isect.t, isNewClosest);
        }
#endif
      }
#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("\n");
      }
#endif
      objectHit = triangleHit && (closest.t >= tNear) && (closest.t <= tFar);
      if (foundLeaf) --stackEnd;
#ifdef DEBUG_TRAVERSE
      if (rayIdx == DEBUG_TRAVERSE_THREAD_ID && foundLeaf) {
        printf("[%08d]", currentId);
        for (int i = 0; i < depth; ++i) printf("  ");
        printf("hit = %d numPrimitives = %d offset = %d t_best = %f\n",
               objectHit, numPrimitives, offset, closest.t);
      }
#endif
    }  // end of while (!(objectHit | stackEmpty))

#ifdef DEBUG_TRAVERSE
    if (rayIdx == DEBUG_TRAVERSE_THREAD_ID) {
      printf("numNodes = %d numLeaves = %d\n", numNodes, numLeaves);
    }
#endif
    hits[rayIdx] = closest;
  } while (true);
}

__global__
#ifdef USE_TRACE_KERNEL_LAUNCH_BOUNDS
    __launch_bounds__(THREADS_PER_BLOCK, MIN_BLOCKS)
#endif
        void traceKernel(const __restrict__ float4* rays,
                         const __restrict__ uint4* nodes,
                         const __restrict__ float4* vertices,
                         const __restrict__ int4* indices,
                         const __restrict__ uint32_t* references,
                         const Aabb4 bounds, uint32_t numTriangles,
                         uint32_t numVertices, uint32_t numRays, Hit* hits) {
  intersectOctree(rays, nodes, vertices, indices, references, bounds,
                  numTriangles, numVertices, numRays, hits);
}

CUDAOctreeRenderer::CUDAOctreeRenderer(const ConfigLoader& c) : config(c) {
  image.filename = config.imageFilename;
  image.width = config.imageWidth;
}

CUDAOctreeRenderer::CUDAOctreeRenderer(const ConfigLoader& c,
                                       const BuildOptions& options)
    : config(c), buildOptions(options) {
  image.filename = config.imageFilename;
  image.width = config.imageWidth;
}

void CUDAOctreeRenderer::createRaysOrtho(Ray** d_rays, int* numRays) {
  float margin = 0.05f;
  int yOffset = 0;
  int yStride = 1;

  float3& bbmax = scene.bbmax;
  float3& bbmin = scene.bbmin;
  float3 bbspan = bbmax - bbmin;

  // Set height according to aspect ratio of bounding box
  image.height = (int)(image.width * bbspan.y / bbspan.x);

  float dx = bbspan.x * (1 + 2 * margin) / image.width;
  float dy = bbspan.y * (1 + 2 * margin) / image.height;
  float x0 = bbmin.x - bbspan.x * margin + dx / 2;
  float y0 = bbmin.y - bbspan.y * margin + dy / 2;
  float z = bbmin.z - std::max(bbspan.z, 1.0f) * .001f;
  int rows = idivCeil((image.height - yOffset), yStride);
  int count = image.width * rows;

  // Allocate buffer for rays.
  CHK_CUDA(hipMalloc(d_rays, sizeof(Ray) * count));

  // Generate rays on device.
  dim3 blockSize(32, 16);
  dim3 gridSize(idivCeil(image.width, blockSize.x),
                idivCeil(rows, blockSize.y));
  std::cout << " width = " << image.width << " height = " << image.height
            << " rows = " << rows << "\n";
  createRaysOrthoKernel<<<gridSize, blockSize>>>(
      image.width, rows, x0, y0 + dy * yOffset, z, dx, dy * yStride,
      (float4*)*d_rays);
  CHK_CUDA(hipDeviceSynchronize());

  *numRays = count;
}

void CUDAOctreeRenderer::loadScene() {
  SceneLoader loader(config.objFilename);
  loader.load(&scene);
}

void CUDAOctreeRenderer::render() {
  int4* d_indices;
  float4* d_vertices;

  // Clear the device.  It is ours now.
  CHK_CUDA(hipDeviceReset());

  loadScene();

  CHK_CUDA(hipMalloc((void**)&d_indices, scene.numTriangles * sizeof(int4)));
  CHK_CUDA(
      hipMalloc((void**)&d_vertices, scene.numTriangles * sizeof(float4)));

  LOG(DEBUG) << "numTriangles = " << scene.numTriangles << " "
             << " numVertices = " << scene.numVertices << "\n";

  int4* indices = scene.indices;

  CHK_CUDA(hipMemcpy(d_indices, indices, scene.numTriangles * sizeof(int4),
                      hipMemcpyHostToDevice));

  float4* vertices = scene.vertices;

  CHK_CUDA(hipMemcpy(d_vertices, vertices, scene.numVertices * sizeof(float4),
                      hipMemcpyHostToDevice));

  traceOnDevice(d_indices, d_vertices);

  CHK_CUDA(hipFree(d_indices));
  CHK_CUDA(hipFree(d_vertices));

  CHK_CUDA(hipDeviceReset());
}

void CUDAOctreeRenderer::buildOnDevice(Octree<LAYOUT_AOS>* d_octree) {}

#define DEBUG_CHECK_FILE_OCTREE
void CUDAOctreeRenderer::buildFromFile(Octree<LAYOUT_AOS>* d_octree) {
  Octree<LAYOUT_AOS> octreeFileAos;
  LOG(DEBUG) << "Building from: " << buildOptions.info << "\n";
  octreeFileAos.buildFromFile(buildOptions.info);
  octreeFileAos.setGeometry(NULL, NULL, scene.numTriangles, scene.numVertices);
  Octree<LAYOUT_AOS> octreeFileSoa;
  octreeFileSoa.copy(octreeFileAos);
  octreeFileSoa.copyToGpu(d_octree);
#ifdef DEBUG_CHECK_FILE_OCTREE
  Octree<LAYOUT_AOS> octreeFileSoaCheck;
  octreeFileSoaCheck.copyFromGpu(d_octree);
  LOG(DEBUG) << octreeFileSoaCheck << "\n";
#endif
}

void CUDAOctreeRenderer::build(Octree<LAYOUT_AOS>* d_octree) {
  switch (buildOptions.type) {
    case BuildOptions::BUILD_FROM_FILE:
      buildFromFile(d_octree);
      break;
    case BuildOptions::BUILD_ON_DEVICE:
      buildOnDevice(d_octree);
      break;
    default:
      break;
  }
}

void CUDAOctreeRenderer::traceOnDevice(int4* indices, float4* vertices) {
  const int numThreadsPerBlock = THREADS_PER_BLOCK;
  const int numWarps = 200;

  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

  //      (numRays + WARP_BATCH_SIZE - 1) / WARP_BATCH_SIZE;
  const int numBlocks = (numWarps + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;

  // Allocate rays.
  CHK_CUDA(hipMalloc(&d_rays, sizeof(Ray) * numRays));

  // Generate rays.
  createRaysOrtho(&d_rays, &numRays);

  // Allocate hits for results.
  CHK_CUDA(hipMalloc(&d_hits, sizeof(Hit) * numRays));
  std::vector<Hit> initialHits(numRays);
  const Hit badHit = {0.0f, -1, 0.0f, 0.0f};

  // Initialize to non-hit.
  for (int i = 0; i < numRays; ++i) initialHits[i] = badHit;
  CHK_CUDA(hipMemcpy(d_hits, &initialHits[0], sizeof(Hit) * numRays,
                      hipMemcpyHostToDevice));

  LOG(DEBUG) << "WARP_LOAD_FACTOR = " << WARP_LOAD_FACTOR
             << " WARPS_PER_BLOCK = " << WARPS_PER_BLOCK
             << " numRays = " << numRays << " numWarps = " << numWarps
             << " numThreadsPerBlock = " << numThreadsPerBlock
             << " numBlocks = " << numBlocks
             << " numThreads = " << numBlocks * THREADS_PER_BLOCK << "\n";

  Octree<LAYOUT_AOS>* d_octree = NULL;
  CHK_CUDA(hipMalloc((void**)(&d_octree), sizeof(Octree<LAYOUT_AOS>)));

  build(d_octree);

#ifdef UPDATE_HITS_SOA
  LOG(DEBUG) << "Using SOA format for hits.\n";
#endif
  LOG(DEBUG) << "Ray tracing...\n";

  // OK, let's bind textures.
  hipBindTexture(0, texture_rays, d_rays, numRays * sizeof(Ray));
  const NodeStorage<LAYOUT_AOS>* d_nodeStorage = d_octree->nodeStoragePtr();
  uint32_t numNodes = 0;
  CHK_CUDA(hipMemcpy(&numNodes, &(d_nodeStorage->numNodes), sizeof(uint32_t),
                      hipMemcpyDeviceToHost));
  LOG(DEBUG) << "numNodes = " << numNodes << "\n";
  uint4* d_nodes;
  CHK_CUDA(hipMemcpy(&d_nodes, &(d_nodeStorage->nodes), sizeof(OctNode128*),
                      hipMemcpyDeviceToHost))
  hipBindTexture(0, texture_nodes, d_nodes, sizeof(OctNode128) * numNodes);
  hipBindTexture(0, texture_vertices, vertices,
                  scene.numVertices * sizeof(float4));
  hipBindTexture(0, texture_indices, indices,
                  scene.numTriangles * sizeof(int4));
  uint32_t numReferences = 0;
  CHK_CUDA(hipMemcpy(&numReferences, d_octree->numTriangleReferencesPtr(),
                      sizeof(uint32_t), hipMemcpyDeviceToHost));
  LOG(DEBUG) << "numReferences = " << numReferences << "\n";
  uint32_t* d_references;
  CHK_CUDA(hipMemcpy(&d_references, d_octree->triangleIndicesPtr(),
                      sizeof(uint32_t*), hipMemcpyDeviceToHost));
  hipBindTexture(0, texture_references, d_references,
                  numReferences * sizeof(int));

  uint32_t nextRay = 0;
  CHK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(nextRayIndex), &nextRay, sizeof(uint32_t)));
  Aabb4 bounds;
  bounds.min = make_float4(scene.bbmin, 0.0);
  bounds.max = make_float4(scene.bbmax, 0.0);
  float time;
  hipEvent_t start_event, stop_event;
#if 0 
  size_t logLimit = 0;
  hipDeviceGetLimit(&logLimit, hipLimitPrintfFifoSize);
  printf("--->Old logLimit = %d\n", logLimit);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 10 * logLimit);
  hipDeviceGetLimit(&logLimit, hipLimitPrintfFifoSize);
  printf("--->New logLimit = %d\n", logLimit);
#endif
  CHK_CUDA(hipEventCreate(&start_event));
  CHK_CUDA(hipEventCreate(&stop_event));
  CHK_CUDA(hipEventRecord(start_event, 0));
  traceKernel<<<numBlocks, numThreadsPerBlock>>>(
      reinterpret_cast<float4*>(d_rays), d_nodes, vertices, indices,
      d_references, bounds, scene.numTriangles, scene.numVertices, numRays,
      d_hits);
  CHK_CUDA(hipEventRecord(stop_event, 0));
  CHK_CUDA(hipEventSynchronize(stop_event));
  hipEventElapsedTime(&time, start_event, stop_event);

  LOG(DEBUG) << "Done...\n";
  LOG(DEBUG) << "Elapsed time = " << time * 1000.0 << " microsec"
             << ", " << time << " millisec\n";
  float ns_per_ray = 1000000.0 * time / numRays;
  LOG(DEBUG) << "Traced " << numRays << " rays at " << ns_per_ray
             << " nanoseconds per ray\n";
  float mrays_sec = numRays / (1000.0 * time);
  LOG(DEBUG) << "Rate is " << mrays_sec << " million rays per second.\n";
  Octree<LAYOUT_AOS>::freeOnGpu(d_octree);
  CHK_CUDA(hipFree((void*)(d_octree)));
#ifdef UPDATE_HITS_SOA
  LOG(DEBUG) << "Converting hits from SOA to AOS.\n";
  reorderHitsKernel<<<numBlocks, numThreadsPerBlock>>>(d_hits, numRays);
  hipDeviceSynchronize();
  LOG(DEBUG) << "SOA to AOS conversion done.\n";
#endif

  // Copy hits locally.
  localHits.resize(numRays);
  CHK_CUDA(hipMemcpy(&localHits[0], d_hits, sizeof(Hit) * numRays,
                      hipMemcpyDeviceToHost));

  CHK_CUDA(hipFree(d_hits));
  CHK_CUDA(hipFree(d_rays));
}

void CUDAOctreeRenderer::shade() {
  image.resize();

  float3 backgroundColor = {0.2f, 0.2f, 0.2f};

  Hit* hits = &localHits[0];

  for (size_t i = 0; i < numRays; i++) {
    if (hits[i].triId < 0) {
      image.pixel[i] = backgroundColor;
#if 0 
      int width = image.width;
      int x = i % width;
      int y = i / width;
      if ((x > 200 && x < 300) && (y > 200 && y < 300)) {
        image.pixel[i] = make_float3(1.0f, 0.0f, 0.0f);
        std::cout << "i = " << i << "\n";
      }
#endif
    } else {
      if (hits[i].triId > scene.numTriangles) {
#if 0
        std::cout << " Got out of bounds triangle ID: " << hits[i].triId << "\n";
#endif
        continue;
      }
      const int4 tri = scene.indices[hits[i].triId];
      const float4 v0 = scene.vertices[tri.x];
      const float4 v1 = scene.vertices[tri.y];
      const float4 v2 = scene.vertices[tri.z];
      const float4 e0 = v1 - v0;
      const float4 e1 = v2 - v0;
      const float3 n = normalize(
          cross(make_float3(e0.x, e0.y, e0.z), make_float3(e1.x, e1.y, e1.z)));

      image.pixel[i] = 0.5f * n + make_float3(0.5f, 0.5f, 0.5f);
    }
  }
}

}  // namespace oct
